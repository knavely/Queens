#include "hip/hip_runtime.h"
#include "generator.cuh"
//#include "hip/hip_runtime.h"
//#include <hiprand.h>
#include "philox.h"


typedef unsigned long long BOARD;

__device__ int geomNV(float x, int id) {
  static int p = 10000;
  thrust::default_random_engine rng;
  thrust::uniform_real_distribution<float> uniform(0.0, 1.0);
  //rng.discard(id);
  //float r = uniform(rng);
  float r = x + 1;
  //   printf("%f\n",r);
  int i = 0;
  while(r > x) {
    rng.discard(id*p++);
    r = uniform(rng);   
    ++i;
  }
  return i;
}

__device__ inline BOARD ROL64(BOARD a, unsigned int offset){
  const int _offset = offset;
  return ((offset != 0) ? ((a << _offset) ^ (a >> (64-offset))) : a);
}

__device__  MBOARD32 genWordNV32(float x, int m, int id) {
  //B = 1, A = 0
  //float c = .01;
  float c = (pow(x,2)*((1-pow(x,m+32))*(1-pow(x,m+32))))/(pow(1-x,2));
 
  MBOARD32 g = {0ULL};
  //int nB = (geomNV(x+.5,id) % (m+1));
  int nB = (geomNV(x,id) % (m+1));
  //  printf("%i\n",nB);
  g.board[0] = g.board[0] | ((1ULL << nB) - 1);
  int nA = (geomNV(x,id) % (1024* m+1));
  int core = geomNV(c,id);
  for(int i = 0; i < core; ++i) {
    int ya = (geomNV(x,id) % ((1024)*m+1)) + 1;
    int yb = ((geomNV(x,id) % (m+1)) +1);

    g = LShift(g,ya);
    g = LShift(g,yb);
    g.board[0] = g.board[0] | ((1ULL << yb) - 1);
    //g.board[((i+nB)/64) % 16] = ROL64(g.board[((i+nB)/64) % 16],(ya + yb));
    //g.board[((i+nB)/64) % 16] = g.board[((i+nB)/64) % 16] | ((1ULL << yb) -1);
  }
  //  printf("%i , %i \n", nB,nA);
  g.board[15] = (g.board[15] << nA) & ~(ROL64(1,nA) -1);
  
  return g;
}

__device__  MBOARD genWordNV(float x, int m, int id) {
  //B = 1, A = 0
  float c = (pow(x,2)*((1-pow(x,m))*(1-pow(x,m))))/(pow(1-x,2));
 
  MBOARD g = {0ULL};
  //  int nB = (geomNV(x+.5,id) % (m+1));
  int nB = (geomNV(x,id) % (m+1));
  //  printf("%i\n",nB);
  g.board[0] = g.board[0] | ((1ULL << nB) - 1);
  /*
  int nA = geomNV(x,id)*4 % (256*m+1);
  int core = geomNV(c,id);
  for(int i = 0; i < core; ++i) {
    int ya = (geomNV(x,id) % (300*m+1))*4 + 1;
    int yb = ((geomNV(x+.5,id) % (m+1)) +1);
    g.board[((i+nB)/64) % 4] = ROL64(g.board[((i+nB)/64) % 4],(ya + yb));
    g.board[((i+nB)/64) % 4] = g.board[((i+nB)/64) % 4] | ((1ULL << yb) -1);
  }
  */
  int nA = geomNV(x,id) % (m+1);
  int core = geomNV(c,id);
  for(int i = 0; i < core; ++i) {
    int ya = (geomNV(x,id) % (300*m+1))*4 + 1;
    int yb = ((geomNV(x,id) % (m+1)) +1);
    g.board[((i+nB)/64) % 4] = ROL64(g.board[((i+nB)/64) % 4],(ya + yb));
    g.board[((i+nB)/64) % 4] = g.board[((i+nB)/64) % 4] | ((1ULL << yb) -1);
  }
  //  printf("%i , %i \n", nB,nA);
  g.board[3] = g.board[3] & ~(ROL64(1,nA) -1);
  
  return g;
}

__device__  int geom(float x,int id) {
  static int p = 0;
  //if(p == 5000)
  p = p + 10000 + id;
  Philox_2x32<20> sampler;
  int i = 0;
  //x = x * 100;
  int r = x+1;
  float rr = x+1;
  while(rr > x) {
    //rr = sampler.rand_int(p+5001,p+5002,p+5003,100);
    rr = sampler.rand_float(p+id,0,1.0);
    p++;
    ++i;
  }
  return i;
}

__device__  BOARD genWord(float x, int m, int id) {
  //B = 1, A = 0
  float c = (pow(x,2)*((1-pow(x,m))*(1-pow(x,m))))/(pow(1-x,2));
  BOARD g = 0ULL;
  int nB = geom(x,id) % (m+1);
  g = g | ((1ULL << nB) - 1);
  
  int nA = geom(x,id) % (m+1);
  int core = geom(c,id);
  for(int i = 0; i < core; ++i) {
    int ya = (geom(x,id) % (m)) + 1;
    int yb = (geom(x,id) % (m)) + 1;
    g = g << (ya + yb);
    g = g | ((1ULL << yb) -1);
  }
  
  //  printf("%i , %i \n", nB,nA);
  g = g << nA;
  return g;
}
  
__device__  MBOARD genMBOARD(float x, int m, int id) {
  return {.board = {genWord(x,m,id),genWord(x,m,id),genWord(x,m,id),genWord(x,m,id)}};
}

__host__ int geomH(float x) {
  int i = 0;
  x = x * 100;
  int r = x+1;
  while(r > x) {
    r = rand() % 100;
    ++i;
  }
  return i;
}
__host__  MBOARD genWordH(float x, int m) {
  //B = 1, A = 0
  float c = (pow(x,2)*((1-pow(x,m))*(1-pow(x,m))))/(pow(1-x,2));
  MBOARD g = {0ULL};
  int nB = geomH(x) % (m+1);
  g.board[0] = g.board[0] | ((1ULL << nB) - 1);
  
  
  int nA = geomH(x) % (256*m+1);
  int core = geomH(c);
  for(int i = 0; i < core; ++i) {
    int ya = (geomH(x) % (256*m)) + 1;
    int yb = (geomH(x) % (m)) + 1;
    g.board[((i+nB)/64) % 4] = g.board[((i+nB)/64) % 4] << (ya + yb);
    g.board[((i+nB)/64) % 4] = g.board[((i+nB)/64) % 4] | ((1ULL << yb) -1);
  }
  
  //  printf("%i , %i \n", nB,nA);
  g.board[3] = g.board[3] & ~((1 << nA) -1);
  
  return g;
}

__host__  MBOARD genMBOARDH(float x, int m) {
  //return {.board = {genWordH(x,m),genWordH(x,m),genWordH(x,m),genWordH(x,m)}};
  return genWordH(x,m);
}
/*int main(){
  srand(time(0));
  genWord(.5,4);
}
*/
