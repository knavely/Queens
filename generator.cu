#include "hip/hip_runtime.h"
#include "generator.h"
#include "hip/hip_runtime.h"
#include <hiprand.h>
#include "philox.h"

typedef unsigned long long BOARD;

__device__  int geom(float x,int id) {
  static int p = 5000;
  if(p == 5000)
    p = p * id;
  //Philox_2x32 sampler;
  int i = 0;
  x = x * 100;
  int r = x+1;
  while(r > x) {
    //r = sampler.rand_int(p,p,p,100);
    p++;
    ++i;
  }
  return i;
}

__device__  BOARD genWord(float x, int m, int id) {
  //B = 1, A = 0
  float c = (pow(x,2)*((1-pow(x,m))*(1-pow(x,m))))/(pow(1-x,2));
  BOARD g = 0ULL;
  int nB = geom(x,id) % (m+1);
  g = g | ((1ULL << nB) - 1);
  
  int nA = geom(x,id) % (m+1);
  int core = geom(c,id);
  for(int i = 0; i < core; ++i) {
    int ya = (geom(x,id) % (m)) + 1;
    int yb = (geom(x,id) % (m)) + 1;
    g = g << (ya + yb);
    g = g | ((1ULL << yb) -1);
  }
  
  //  printf("%i , %i \n", nB,nA);
  g = g << nA;
  return g;
}

__device__  MBOARD genMBOARD(float x, int m, int id) {
  return {.board = {genWord(x,m,id),genWord(x,m,id),genWord(x,m,id),genWord(x,m,id)}};
}

__host__ int geomH(float x) {
  int i = 0;
  x = x * 100;
  int r = x+1;
  while(r > x) {
    r = rand() % 100;
    ++i;
  }
  return i;
}
__host__  MBOARD genWordH(float x, int m) {
  //B = 1, A = 0
  float c = (pow(x,2)*((1-pow(x,m))*(1-pow(x,m))))/(pow(1-x,2));
  MBOARD g = {0ULL};
  int nB = geomH(x) % (m+1);
  g.board[0] = g.board[0] | ((1ULL << nB) - 1);
  
  
  int nA = geomH(x) % (m+1);
  int core = geomH(c);
  for(int i = 0; i < core; ++i) {
    int ya = (geomH(x) % (m)) + 1;
    int yb = (geomH(x) % (m)) + 1;
    g.board[((i+nB)/64) % 4] = g.board[((i+nB)/64) % 4] << (ya + yb);
    g.board[((i+nB)/64) % 4] = g.board[((i+nB)/64) % 4] | ((1ULL << yb) -1);
  }
  
  //  printf("%i , %i \n", nB,nA);
  g.board[3] = g.board[3] & ~((1 << nA) -1);
  
  return g;
}

__host__  MBOARD genMBOARDH(float x, int m) {
  //return {.board = {genWordH(x,m),genWordH(x,m),genWordH(x,m),genWordH(x,m)}};
  return genWordH(x,m);
}
/*int main(){
  srand(time(0));
  genWord(.5,4);
}
*/
