#include "hip/hip_runtime.h"

#include "generator.cuh"
//#include "hip/hip_runtime.h"
//#include <hiprand.h>
#include "philox.h"


typedef unsigned long long BOARD;

__device__ int geomNV(float x, int id) {
  thrust::default_random_engine rng;
  thrust::uniform_real_distribution<float> uniform(0.0, 1.0);
  rng.discard(5000*id);
  float r = uniform(rng);
  //   printf("%f\n",r);
  int i = 0;
  while(r > x) {
     rng.discard(5000*id);
    r = uniform(rng);
    ++i;
  }
  return i;
}

__device__ inline BOARD ROL64(BOARD a, unsigned int offset){
  const int _offset = offset;
  return ((offset != 0) ? ((a << _offset) ^ (a >> (64-offset))) : a);
}

__device__  MBOARD genWordNV(float x, int m, int id) {
  //B = 1, A = 0
  float c = (pow(x,2)*((1-pow(x,m))*(1-pow(x,m))))/(pow(1-x,2));
  MBOARD g = {0ULL};
  int nB = geom(x,id) % (m);
  //  printf("%i\n",nB);
  g.board[0] = g.board[0] | ((1ULL << nB) - 1);
  
  
  int nA = geom(x,id) % (m*256+1);
  int core = geom(c,id);
  for(int i = 0; i < core; ++i) {
    int ya = (geom(x,id) % (m*256)) + 1;
    int yb = (geom(x,id) % (m)) + 1;
    g.board[((i+nB)/64) % 4] = ROL64(g.board[((i+nB)/64) % 4],(ya + yb));
    g.board[((i+nB)/64) % 4] = g.board[((i+nB)/64) % 4] | ((1ULL << yb) -1);
  }
  
  //  printf("%i , %i \n", nB,nA);
  g.board[3] = g.board[3] & ~(ROL64(1,nA) -1);
  
  return g;
}

__device__  int geom(float x,int id) {
  static int p = 5000;
  //if(p == 5000)
  p = p + id;
  Philox_2x32<100> sampler;
  int i = 0;
  x = x * 100;
  int r = x+1;
  while(r > x) {
    r = sampler.rand_int(p,p,p,100);
    p++;
    ++i;
  }
  return i;
}

__device__  BOARD genWord(float x, int m, int id) {
  //B = 1, A = 0
  float c = (pow(x,2)*((1-pow(x,m))*(1-pow(x,m))))/(pow(1-x,2));
  BOARD g = 0ULL;
  int nB = geom(x,id) % (m+1);
  g = g | ((1ULL << nB) - 1);
  
  int nA = geom(x,id) % (m+1);
  int core = geom(c,id);
  for(int i = 0; i < core; ++i) {
    int ya = (geom(x,id) % (m)) + 1;
    int yb = (geom(x,id) % (m)) + 1;
    g = g << (ya + yb);
    g = g | ((1ULL << yb) -1);
  }
  
  //  printf("%i , %i \n", nB,nA);
  g = g << nA;
  return g;
}
  
__device__  MBOARD genMBOARD(float x, int m, int id) {
  return {.board = {genWord(x,m,id),genWord(x,m,id),genWord(x,m,id),genWord(x,m,id)}};
}

__host__ int geomH(float x) {
  int i = 0;
  x = x * 100;
  int r = x+1;
  while(r > x) {
    r = rand() % 100;
    ++i;
  }
  return i;
}
__host__  MBOARD genWordH(float x, int m) {
  //B = 1, A = 0
  float c = (pow(x,2)*((1-pow(x,m))*(1-pow(x,m))))/(pow(1-x,2));
  MBOARD g = {0ULL};
  int nB = geomH(x) % (m+1);
  g.board[0] = g.board[0] | ((1ULL << nB) - 1);
  
  
  int nA = geomH(x) % (256*m+1);
  int core = geomH(c);
  for(int i = 0; i < core; ++i) {
    int ya = (geomH(x) % (256*m)) + 1;
    int yb = (geomH(x) % (m)) + 1;
    g.board[((i+nB)/64) % 4] = g.board[((i+nB)/64) % 4] << (ya + yb);
    g.board[((i+nB)/64) % 4] = g.board[((i+nB)/64) % 4] | ((1ULL << yb) -1);
  }
  
  //  printf("%i , %i \n", nB,nA);
  g.board[3] = g.board[3] & ~((1 << nA) -1);
  
  return g;
}

__host__  MBOARD genMBOARDH(float x, int m) {
  //return {.board = {genWordH(x,m),genWordH(x,m),genWordH(x,m),genWordH(x,m)}};
  return genWordH(x,m);
}
/*int main(){
  srand(time(0));
  genWord(.5,4);
}
*/
