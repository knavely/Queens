#include "hip/hip_runtime.h"
#include "generator.cuh"

__device__ __host__ BOARD getBit(MBOARD32 B, BOARD b) {
  BOARD d = b/64;
  BOARD r  = b % 64;
  return ((1ULL << r) & B.board[d]);
}

__device__ __host__ MBOARD32 bishopDiagonal1() {
  BOARD B1 = 1ULL | 1ULL << 33;
  BOARD B2 = B1 << 2;
  BOARD B3 = B2 << 2;
  BOARD B4 = B3 << 2;
  BOARD B5 = B4 << 2;
  BOARD B6 = B5 << 2;
  BOARD B7 = B6 << 2;
  BOARD B8 = B7 << 2;
  BOARD B9 = B8 << 2;
  BOARD B10 = B9 << 2;
  BOARD B11 = B10 << 2;
  BOARD B12 = B11 << 2;
  BOARD B13 = B12 << 2;
  BOARD B14 = B13 << 2;
  BOARD B15 = B14 << 2;
  BOARD B16 = B15 << 2;

  return (MBOARD32){.board = {B1,B2,B3,B4,B5,B6,B7,B8,B9,B10,B11,B12,B13,B14,B15,B16}};
}

__device__ __host__ MBOARD32 bishopDiagonal2() {
  BOARD B1 = 1ULL << 31 | 1ULL << 62;
  BOARD B2 = B1 >> 2;
  BOARD B3 = B2 >> 2;
  BOARD B4 = B3 >> 2;
  BOARD B5 = B4 >> 2;
  BOARD B6 = B5 >> 2;
  BOARD B7 = B6 >> 2;
  BOARD B8 = B7 >> 2;
  BOARD B9 = B8 >> 2;
  BOARD B10 = B9 >> 2;
  BOARD B11 = B10 >> 2;
  BOARD B12 = B11 >> 2;
  BOARD B13 = B12 >> 2;
  BOARD B14 = B13 >> 2;
  BOARD B15 = B14 >> 2;
  BOARD B16 = B15 >> 2;

  return (MBOARD32){.board = {B1,B2,B3,B4,B5,B6,B7,B8,B9,B10,B11,B12,B13,B14,B15,B16}};
}

__device__ __host__ MBOARD32 compliment(MBOARD32 B) {
  MBOARD32 C = B;
  for(int i = 0; i < 16; ++i) {
    C.board[i] = ~B.board[i];
  }
  return C;
}

__device__ __host__ MBOARD32 LShift(MBOARD32 B,int k) {
  BOARD t = 0; //(B.board[0] & (((1ULL << (k)) - 1) << 64-k));
  for(int i = 0; i < 16; ++i) {
    BOARD T = B.board[i];
    B.board[i] = ((B.board[i] << k) | t);
    t = (T & (((1ULL << (k)) - 1) << 64-k)) >> 64-k;	
  }
  return B;
}

__device__ __host__ MBOARD32 RShift(MBOARD32 B,int k) {
  BOARD t = 0ULL; //B.board[0] & (((1ULL << k) - 1) << 64-k);
  for(int i = 16-1; i >= 0; --i) {
    BOARD T = B.board[i];
    B.board[i] = ((B.board[i] >> k) | t);
    t = (T & ((1ULL << (k)) - 1)) << 64-k;	
  }
  return B;
}

__device__ __host__ MBOARD32 RShiftBishop1(MBOARD32 mb, int k) {
  MBOARD32 r = mb;
  for(int i = 0; i < k; ++i) {
    r = RShift(r,1);
    int p = 31; 
    r.board[0] = ~(1ULL << p | 1ULL << (p+32)) & r.board[0];
    r.board[1] = ~(1ULL << p | 1ULL << (p+32)) & r.board[1];
    r.board[2] = ~(1ULL << p | 1ULL << (p+32)) & r.board[2];
    r.board[3] = ~(1ULL << p | 1ULL << (p+32)) & r.board[3];
    r.board[4] = ~(1ULL << p | 1ULL << (p+32)) & r.board[4];
    r.board[5] = ~(1ULL << p | 1ULL << (p+32)) & r.board[5];
    r.board[6] = ~(1ULL << p | 1ULL << (p+32)) & r.board[6];
    r.board[7] = ~(1ULL << p | 1ULL << (p+32)) & r.board[7];
    r.board[8] = ~(1ULL << p | 1ULL << (p+32)) & r.board[8];
    r.board[9] = ~(1ULL << p | 1ULL << (p+32)) & r.board[9];
    r.board[10] = ~(1ULL << p | 1ULL << (p+32)) & r.board[10];
    r.board[11] = ~(1ULL << p | 1ULL << (p+32)) & r.board[11];
    r.board[12] = ~(1ULL << p | 1ULL << (p+32)) & r.board[12];
    r.board[13] = ~(1ULL << p | 1ULL << (p+32)) & r.board[13];
    r.board[14] = ~(1ULL << p | 1ULL << (p+32)) & r.board[14];
    r.board[15] = ~(1ULL << p | 1ULL << (p+32)) & r.board[15];

  }
  return r;
}

__device__ __host__ MBOARD32 LShiftBishop1(MBOARD32 mb, int k) {
  MBOARD32 l = mb;
  for(int i = 0; i < k; ++i) {
    l = LShift(l,1);
    int p = 0;
    l.board[0] = ~(1ULL << p | 1ULL << (p+32)) & l.board[0];
    l.board[1] = ~(1ULL << p | 1ULL << (p+32)) & l.board[1];
    l.board[2] = ~(1ULL << p | 1ULL << (p+32)) & l.board[2];
    l.board[3] = ~(1ULL << p | 1ULL << (p+32)) & l.board[3];
    l.board[4] = ~(1ULL << p | 1ULL << (p+32)) & l.board[4];
    l.board[5] = ~(1ULL << p | 1ULL << (p+32)) & l.board[5];
    l.board[6] = ~(1ULL << p | 1ULL << (p+32)) & l.board[6];
    l.board[7] = ~(1ULL << p | 1ULL << (p+32)) & l.board[7];
    l.board[8] = ~(1ULL << p | 1ULL << (p+32)) & l.board[8];
    l.board[9] = ~(1ULL << p | 1ULL << (p+32)) & l.board[9];
    l.board[10] = ~(1ULL << p | 1ULL << (p+32)) & l.board[10];
    l.board[11] = ~(1ULL << p | 1ULL << (p+32)) & l.board[11];
    l.board[12] = ~(1ULL << p | 1ULL << (p+32)) & l.board[12];
    l.board[13] = ~(1ULL << p | 1ULL << (p+32)) & l.board[13];
    l.board[14] = ~(1ULL << p | 1ULL << (p+32)) & l.board[14];
    l.board[15] = ~(1ULL << p | 1ULL << (p+32)) & l.board[15];

  }
  return l;
}

__device__ __host__ MBOARD32 LShiftRook(MBOARD32 mb, int k) {
  MBOARD32 d = mb;
  for(int i = 0; i < k; ++i) {
    d = LShift(d,32);
  }
  return d;
}

__device__ __host__ MBOARD32 RShiftRook(MBOARD32 mb, int k) {
  MBOARD32 d = mb;
  for(int i = 0; i < k; ++i) {
    d = RShift(d,32);
  }
  return d;
}

__device__ __host__ MBOARD32 UShiftRook(MBOARD32 mb, int k) {
  MBOARD32 d = mb;
  for(int i = 0; i < k; ++i) {
    d = LShift(d,1);
  }
  return d;
}

__device__ __host__ MBOARD32 DShiftRook(MBOARD32 mb, int k) {
  MBOARD32 d = mb;
  for(int i = 0; i < k; ++i) {
    d = RShift(d,1);
  }
  return d;
}

__device__ __host__ void drawBoard(MBOARD32 white, MBOARD32 black) {
  for(int r = 0; r < 32; ++r) {
    for(int c = 0; c < 32; ++c) {
      if(getBit(white, r*32 + c)) 
	printf(" %s ", "\u2655");
      else if(getBit(black,r*32 + c)) 
	printf(" %s ", "\u265B");
      else{  
	printf(" %s ", "\u25A0");
      }
    }
    printf("\n");
  }
  printf("\n");  
}

__device__ __host__ BOARD Positive(MBOARD32 B) {
  return (BOARD)(B.board[0] + B.board[1] + B.board[2] + B.board[3] + B.board[4] + B.board[5] + B.board[6] + B.board[7]
		 + B.board[8] + B.board[9] + B.board[10] + B.board[11] + B.board[12] + B.board[13] + B.board[14] + B.board[15]);
  }

__device__ __host__ MBOARD32 And(MBOARD32 A, MBOARD32 B) {
  return (MBOARD32){.board = {(A.board[0] & B.board[0]),(A.board[1] & B.board[1]),
      (A.board[2] & B.board[2]), (A.board[3] & B.board[3]),
      (A.board[4] & B.board[4]),(A.board[5] & B.board[5]),
      (A.board[6] & B.board[6]), (A.board[7] & B.board[7]),
      (A.board[8] & B.board[8]),(A.board[9] & B.board[9]),
      (A.board[10] & B.board[10]), (A.board[11] & B.board[11]),
      (A.board[12] & B.board[12]),(A.board[13] & B.board[13]),
      (A.board[14] & B.board[14]), (A.board[15] & B.board[15])}};
}
__device__ __host__ const MBOARD32 Or(const MBOARD32 A, const MBOARD32 B) {
  return (MBOARD32){.board = {
      A.board[0] | B.board[0],
      A.board[1] | B.board[1],
      A.board[2] | B.board[2],
      A.board[3] | B.board[3],
      A.board[4] | B.board[4],
      A.board[5] | B.board[5],
      A.board[6] | B.board[6],
      A.board[7] | B.board[7],
      A.board[8] | B.board[8],
      A.board[9] | B.board[9],
      A.board[10] | B.board[10],
      A.board[11] | B.board[11],
      A.board[12] | B.board[12],
      A.board[13] | B.board[13],
      A.board[14] | B.board[14],
      A.board[15] | B.board[15]}};
}

__device__ __host__ MBOARD32 Not(MBOARD32 B){
  return (MBOARD32){.board = {~(B.board[0]),~(B.board[1]),~(B.board[2]),~(B.board[3]),~(B.board[4]),~(B.board[5]),~(B.board[6]),~(B.board[7]),
    ~(B.board[8]),~(B.board[9]),~(B.board[10]),~(B.board[11]),~(B.board[12]),~(B.board[13]),~(B.board[14]),~(B.board[15])}};
}

__device__ __host__ MBOARD32 rookRowMask(){
  return {.board = {0xFFFFFFFFULL,0ULL,0ULL,0ULL,0ULL,0ULL,0ULL,0ULL,0ULL,0ULL,0ULL,0ULL,0ULL,0ULL,0ULL,0ULL}};
}

__device__ __host__ MBOARD32 rookColMask(){
  BOARD pattern = 1ULL | (1ULL << 32);
  return {.board = {pattern,pattern,pattern,pattern,pattern,pattern,pattern,pattern,pattern,pattern,pattern,pattern,pattern,pattern,pattern,pattern}};
}


__device__ __host__ MBOARD32 getRookMask(MBOARD32 queens) {
  MBOARD32 r1 = {0},r2 = {0},r3 = {0},r4 = {0},r5 = {0},r6 = {0},r7 = {0},r8 = {0},l1 = {0},l2 = {0},l3= {0},l4 = {0},l5 = {0},
    l6 = {0},l7 = {0},l8 = {0};
  MBOARD32 u1 = {0},u2 = {0},u3 = {0},u4 = {0},u5 = {0},u6 = {0},u7 = {0},u8 = {0},d1 = {0},d2 = {0},d3 = {0},d4 = {0},d5 = {0}
    ,d6 = {0},d7 = {0},d8 = {0};
  MBOARD32 r9 = {0},r10 = {0},r11 = {0},r12 = {0},r13 = {0},r14 = {0},r15 = {0},r16 = {0},l9 = {0},l10 = {0},l11= {0},l12 = {0},l13 = {0},
    l14 = {0},l15 = {0},l16 = {0};
  MBOARD32 u9 = {0},u10 = {0},u11 = {0},u12 = {0},u13 = {0},u14 = {0},u15 = {0},u16 = {0},d9 = {0},d10 = {0},d11 = {0},d12 = {0},d13 = {0},d14 = {0},d15 = {0},d16 = {0};

  MBOARD32 r17 = {0},r18 = {0},r19 = {0},r20 = {0},r21 = {0},r22 = {0},r23 = {0},r24 = {0},l17 = {0},l18 = {0},l19= {0},l20 = {0},l21 = {0},
    l22 = {0},l23 = {0},l24 = {0};
  MBOARD32 u17 = {0},u18 = {0},u19 = {0},u20 = {0},u21 = {0},u22 = {0},u23 = {0},u24 = {0},d17 = {0},d18 = {0},d19 = {0},d20 = {0},d21 = {0}
    ,d22 = {0},d23 = {0},d24 = {0};
  MBOARD32 r25 = {0},r26 = {0},r27 = {0},r28 = {0},r29 = {0},r30 = {0}, r31 = {0}, r32 = {0},l25 = {0},l26 = {0},l27= {0},l28 = {0},l29 = {0},
    l30 = {0},l31 = {0},l32 = {0};
  MBOARD32 u25 = {0},u26 = {0},u27 = {0},u28 = {0},u29 = {0},u30 = {0},u31 = {0},u32 = {0},d25 = {0},d26 = {0},d27 = {0},d28 = {0},d29 = {0},d30 = {0},d31 = {0},d32 = {0};

  MBOARD32 pos = queens;
  for(MBOARD32 mask = rookRowMask(); Positive(mask); mask = LShiftRook(mask,1)) {
    if(Positive(And(queens, mask))) {
      r1 = Or(r1, And(LShift(queens,1), And(Not(pos),mask)));
      r2 = Or(r2, And(LShift(r1,1), And(Not(pos),mask)));
      r3 = Or(r3, And(LShift(r2,1), And(Not(pos),mask)));
      r4 = Or(r4, And(LShift(r3,1), And(Not(pos),mask)));
      r5 = Or(r5, And(LShift(r4,1), And(Not(pos),mask)));
      r6 = Or(r6, And(LShift(r5,1), And(Not(pos),mask)));
      r7 = Or(r7, And(LShift(r6,1), And(Not(pos),mask)));
      r8 = Or(r8, And(LShift(r7,1), And(Not(pos),mask)));
      r9 = Or(r9, And(LShift(r8,1), And(Not(pos),mask)));
      r10 = Or(r10, And(LShift(r9,1), And(Not(pos),mask)));
      r11 = Or(r11, And(LShift(r10,1), And(Not(pos),mask)));
      r12 = Or(r12, And(LShift(r11,1), And(Not(pos),mask)));
      r13 = Or(r13, And(LShift(r12,1), And(Not(pos),mask)));
      r14 = Or(r14, And(LShift(r13,1), And(Not(pos),mask)));
      r15 = Or(r15, And(LShift(r14,1), And(Not(pos),mask)));
      r16 = Or(r15, And(LShift(r15,1), And(Not(pos),mask)));
      r17 = Or(r17, And(LShift(r16,1), And(Not(pos),mask)));
      r18 = Or(r18, And(LShift(r17,1), And(Not(pos),mask)));
      r19 = Or(r19, And(LShift(r18,1), And(Not(pos),mask)));
      r20 = Or(r20, And(LShift(r19,1), And(Not(pos),mask)));
      r21 = Or(r21, And(LShift(r20,1), And(Not(pos),mask)));
      r22 = Or(r22, And(LShift(r21,1), And(Not(pos),mask)));
      r23 = Or(r23, And(LShift(r22,1), And(Not(pos),mask)));
      r24 = Or(r24, And(LShift(r23,1), And(Not(pos),mask)));
      r25 = Or(r25, And(LShift(r24,1), And(Not(pos),mask)));
      r26 = Or(r26, And(LShift(r25,1), And(Not(pos),mask)));
      r27 = Or(r27, And(LShift(r26,1), And(Not(pos),mask)));
      r28 = Or(r28, And(LShift(r27,1), And(Not(pos),mask)));
      r29 = Or(r29, And(LShift(r28,1), And(Not(pos),mask)));
      r30 = Or(r30, And(LShift(r29,1), And(Not(pos),mask)));
      r31 = Or(r31, And(LShift(r30,1), And(Not(pos),mask)));
      r32 = Or(r32, And(LShift(r31,1), And(Not(pos),mask)));
      
      l1 = Or(l1, And(RShift(queens,1), And(Not(pos),mask)));
      l2 = Or(l2, And(RShift(l1,1), And(Not(pos),mask)));
      l3 = Or(l3, And(RShift(l2,1), And(Not(pos),mask)));
      l4 = Or(l4, And(RShift(l3,1), And(Not(pos),mask)));
      l5 = Or(l5, And(RShift(l4,1), And(Not(pos),mask)));
      l6 = Or(l6, And(RShift(l5,1), And(Not(pos),mask)));
      l7 = Or(l7, And(RShift(l6,1), And(Not(pos),mask)));
      l8 = Or(l8, And(RShift(l7,1), And(Not(pos),mask)));
      l9 = Or(l9, And(RShift(l8,1), And(Not(pos),mask)));
      l10 = Or(l10, And(RShift(l9,1), And(Not(pos),mask)));
      l11 = Or(l11, And(RShift(l10,1), And(Not(pos),mask)));
      l12 = Or(l12, And(RShift(l11,1), And(Not(pos),mask)));
      l13 = Or(l13, And(RShift(l12,1), And(Not(pos),mask)));
      l14 = Or(l14, And(RShift(l13,1), And(Not(pos),mask)));
      l15 = Or(l15, And(RShift(l14,1), And(Not(pos),mask)));
      l16 = Or(l15, And(RShift(l15,1), And(Not(pos),mask)));
      l17 = Or(l17, And(RShift(l16,1), And(Not(pos),mask)));
      l18 = Or(l18, And(RShift(l17,1), And(Not(pos),mask)));
      l19 = Or(l19, And(RShift(l18,1), And(Not(pos),mask)));
      l20 = Or(l20, And(RShift(l19,1), And(Not(pos),mask)));
      l21 = Or(l21, And(RShift(l20,1), And(Not(pos),mask)));
      l22 = Or(l22, And(RShift(l21,1), And(Not(pos),mask)));
      l23 = Or(l23, And(RShift(l22,1), And(Not(pos),mask)));
      l24 = Or(l24, And(RShift(l23,1), And(Not(pos),mask)));
      l25 = Or(l25, And(RShift(l24,1), And(Not(pos),mask)));
      l26 = Or(l26, And(RShift(l25,1), And(Not(pos),mask)));
      l27 = Or(l27, And(RShift(l26,1), And(Not(pos),mask)));
      l28 = Or(l28, And(RShift(l27,1), And(Not(pos),mask)));
      l29 = Or(l29, And(RShift(l28,1), And(Not(pos),mask)));
      l30 = Or(l30, And(RShift(l29,1), And(Not(pos),mask)));
      l31 = Or(l31, And(RShift(l30,1), And(Not(pos),mask)));
      l32 = Or(l32, And(RShift(l31,1), And(Not(pos),mask)));    
    }
  } 
  // BOARD pattern = 1ULL | (1ULL << 8) | (1ULL << 16) | (1ULL << 24)
  //| (1ULL << 32ULL) | (1ULL << 40ULL) | (1ULL << 48ULL) | (1ULL << 56ULL);
  for(MBOARD32 mask = rookColMask(); Positive(mask); mask = LShift(mask,1)) {
      if(Positive(And(queens,mask))) {
      u1 = Or(u1, And(LShift(queens,32), And(Not(pos),mask))); //u1 |= (queens << 8) & ~pos & mask;
      u2 = Or(u2, And(LShift(u1,32), And(Not(pos),mask)));
      u3 = Or(u3, And(LShift(u2,32), And(Not(pos),mask)));
      u4 = Or(u4, And(LShift(u3,32), And(Not(pos),mask)));
      u5 = Or(u5, And(LShift(u4,32), And(Not(pos),mask)));
      u6 = Or(u6, And(LShift(u5,32), And(Not(pos),mask)));
      u7 = Or(u7, And(LShift(u6,32), And(Not(pos),mask)));
      u8 = Or(u8, And(LShift(u7,32), And(Not(pos),mask)));
      u9 = Or(u9, And(LShift(u8,32), And(Not(pos),mask)));
      u10 = Or(u10, And(LShift(u9,32), And(Not(pos),mask)));
      u11 = Or(u11, And(LShift(u10,32), And(Not(pos),mask)));
      u12 = Or(u12, And(LShift(u11,32), And(Not(pos),mask)));
      u13 = Or(u13, And(LShift(u12,32), And(Not(pos),mask)));
      u14 = Or(u14, And(LShift(u13,32), And(Not(pos),mask)));
      u15 = Or(u15, And(LShift(u14,32), And(Not(pos),mask)));
      u16 = Or(u15, And(LShift(u15,32), And(Not(pos),mask)));
      u17 = Or(u17, And(LShift(u16,32), And(Not(pos),mask))); //u1 |= (queens << 8) & ~pos & mask;
      u18 = Or(u18, And(LShift(u17,32), And(Not(pos),mask)));
      u19 = Or(u19, And(LShift(u18,32), And(Not(pos),mask)));
      u20 = Or(u20, And(LShift(u19,32), And(Not(pos),mask)));
      u21 = Or(u21, And(LShift(u20,32), And(Not(pos),mask)));
      u22 = Or(u22, And(LShift(u21,32), And(Not(pos),mask)));
      u23 = Or(u23, And(LShift(u22,32), And(Not(pos),mask)));
      u24 = Or(u24, And(LShift(u23,32), And(Not(pos),mask)));
      u25 = Or(u25, And(LShift(u24,32), And(Not(pos),mask)));
      u26 = Or(u26, And(LShift(u25,32), And(Not(pos),mask)));
      u27 = Or(u27, And(LShift(u26,32), And(Not(pos),mask)));
      u28 = Or(u28, And(LShift(u27,32), And(Not(pos),mask)));
      u29 = Or(u29, And(LShift(u28,32), And(Not(pos),mask)));
      u30 = Or(u30, And(LShift(u29,32), And(Not(pos),mask)));
      u31 = Or(u31, And(LShift(u30,32), And(Not(pos),mask)));
      u32 = Or(u32, And(LShift(u31,32), And(Not(pos),mask)));
      
      d1 = Or(d1, And(RShift(queens,32), And(Not(pos),mask)));
      d2 = Or(d2, And(RShift(d1,32), And(Not(pos),mask)));
      d3 = Or(d3, And(RShift(d2,32), And(Not(pos),mask)));
      d4 = Or(d4, And(RShift(d3,32), And(Not(pos),mask)));
      d5 = Or(d5, And(RShift(d4,32), And(Not(pos),mask)));
      d6 = Or(d6, And(RShift(d5,32), And(Not(pos),mask)));
      d7 = Or(d7, And(RShift(d6,32), And(Not(pos),mask)));
      d8 = Or(d8, And(RShift(d7,32), And(Not(pos),mask)));
      d9 = Or(d9, And(RShift(d8,32), And(Not(pos),mask)));
      d10 = Or(d10, And(RShift(d9,32), And(Not(pos),mask)));
      d11 = Or(d11, And(RShift(d10,32), And(Not(pos),mask)));
      d12 = Or(d12, And(RShift(d11,32), And(Not(pos),mask)));
      d13 = Or(d13, And(RShift(d12,32), And(Not(pos),mask)));
      d14 = Or(d14, And(RShift(d13,32), And(Not(pos),mask)));
      d15 = Or(d15, And(RShift(d14,32), And(Not(pos),mask)));
      d16 = Or(d15, And(RShift(d15,32), And(Not(pos),mask)));
      d17 = Or(d17, And(RShift(d16,32), And(Not(pos),mask)));
      d18 = Or(d18, And(RShift(d17,32), And(Not(pos),mask)));
      d19 = Or(d19, And(RShift(d18,32), And(Not(pos),mask)));
      d20 = Or(d20, And(RShift(d19,32), And(Not(pos),mask)));
      d21 = Or(d21, And(RShift(d20,32), And(Not(pos),mask)));
      d22 = Or(d22, And(RShift(d21,32), And(Not(pos),mask)));
      d23 = Or(d23, And(RShift(d22,32), And(Not(pos),mask)));
      d24 = Or(d24, And(RShift(d23,32), And(Not(pos),mask)));
      d25 = Or(d25, And(RShift(d24,32), And(Not(pos),mask)));
      d26 = Or(d26, And(RShift(d25,32), And(Not(pos),mask)));
      d27 = Or(d27, And(RShift(d26,32), And(Not(pos),mask)));
      d28 = Or(d27, And(RShift(d27,32), And(Not(pos),mask)));
      d29 = Or(d28, And(RShift(d28,32), And(Not(pos),mask)));
      d30 = Or(d30, And(RShift(d29,32), And(Not(pos),mask)));
      d31 = Or(d31, And(RShift(d30,32), And(Not(pos),mask)));
      d32 = Or(d32, And(RShift(d31,32), And(Not(pos),mask)));

    } 
  }
  MBOARD32 rookMask1 =  Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(r1, r2),r3),r4),r5),r6),r7),r8),l1),
											     l2),l3),l4),l5), l6), l7), l8),
									u1), u2), u3), u4), u5), u6), u7), u8),d1), d2), d3), d4), d5), d6), d7), d8);


  MBOARD32 rookMask2 =  Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(r9, r10),r11),r12),r13),r14),r15),r16),l9),
											     l10),l11),l12),l13), l14), l15), l16),
									u9), u10), u11), u12), u13), u14), u15), u16),d9), d10), d11), d12), d13), d14), d15), d16);

  MBOARD32 rookMask3 =  Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(r17, r18),r19),r20),r21),r22),r23),r24),l17),
											     l18),l19),l20),l21), l22), l23), l24),
									u17), u18), u19), u20), u21), u22), u23), u24),d17), d18), d19), d20), d21), d22), d23), d24);

  MBOARD32 rookMask4 =  Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(r25, r26),r27),r28),r29),r30),r31),r32),l25),
											     l26),l27),l28),l29), l30), l31), l32),
									u25), u26), u27), u28), u29), u30), u31), u32),d25), d26), d27), d28), d29), d30), d31), d32);


  return Or(Or(queens,Or(rookMask1, rookMask2)),Or(queens,Or(rookMask3, rookMask4)));	

} 

__device__ __host__ MBOARD32 getBishopMask(MBOARD32 queens) {
  MBOARD32 BISHOP1 = bishopDiagonal1(); //0x8040201008040201;
  MBOARD32 BISHOP2 = bishopDiagonal2(); //0x0102040810204080;  
  MBOARD32 pos = queens;
  MBOARD32 r1 = {0},r2 = {0},r3 = {0},r4 = {0},r5 = {0},r6 = {0},r7 = {0},r8 = {0},l1 = {0},l2 = {0},l3= {0},l4 = {0},l5 = {0},
    l6 = {0},l7 = {0},l8 = {0};
  MBOARD32 u1 = {0},u2 = {0},u3 = {0},u4 = {0},u5 = {0},u6 = {0},u7 = {0},u8 = {0},d1 = {0},d2 = {0},d3 = {0},d4 = {0},d5 = {0}
    ,d6 = {0},d7 = {0},d8 = {0};
  MBOARD32 r9 = {0},r10 = {0},r11 = {0},r12 = {0},r13 = {0},r14 = {0},r15 = {0},r16 = {0},l9 = {0},l10 = {0},l11= {0},l12 = {0},l13 = {0},
    l14 = {0},l15 = {0},l16 = {0};
  MBOARD32 u9 = {0},u10 = {0},u11 = {0},u12 = {0},u13 = {0},u14 = {0},u15 = {0},u16 = {0},d9 = {0},d10 = {0},d11 = {0},d12 = {0},d13 = {0},d14 = {0},d15 = {0},d16 = {0};

  MBOARD32 r17 = {0},r18 = {0},r19 = {0},r20 = {0},r21 = {0},r22 = {0},r23 = {0},r24 = {0},l17 = {0},l18 = {0},l19= {0},l20 = {0},l21 = {0},
    l22 = {0},l23 = {0},l24 = {0};
  MBOARD32 u17 = {0},u18 = {0},u19 = {0},u20 = {0},u21 = {0},u22 = {0},u23 = {0},u24 = {0},d17 = {0},d18 = {0},d19 = {0},d20 = {0},d21 = {0}
    ,d22 = {0},d23 = {0},d24 = {0};
  MBOARD32 r25 = {0},r26 = {0},r27 = {0},r28 = {0},r29 = {0},r30 = {0},r31 = {0},r32 = {0},l25 = {0},l26 = {0},l27= {0},l28 = {0},l29 = {0},
    l30 = {0},l31 = {0},l32 = {0};
  MBOARD32 u25 = {0},u26 = {0},u27 = {0}, u28 = {0},u29 = {0},u30 = {0},u31 = {0},u32 = {0},d25 = {0},d26 = {0},d27 = {0},d28 = {0},d29 = {0},d30 = {0},d31 = {0},d32 = {0};

  for(MBOARD32 mask1 = BISHOP1, mask2 = BISHOP1; Positive(mask2); mask1 = RShiftBishop1(mask1,1), mask2 = LShiftBishop1(mask2,1)) {   
    if(Positive(And(queens,mask2))) {
      //drawBoard(queens,mask2);
      //drawBoard(queens,r5);
      r1 = Or(r1, And(LShift(queens, 33), And(Not(pos),mask2)));
      r2 = Or(r2, And(LShift(r1, 33), And(Not(pos),mask2)));
      r3 = Or(r3, And(LShift(r2, 33), And(Not(pos),mask2)));
      r4 = Or(r4, And(LShift(r3, 33), And(Not(pos),mask2)));
      r5 = Or(r5, And(LShift(r4, 33), And(Not(pos),mask2)));
      r6 = Or(r6, And(LShift(r5, 33), And(Not(pos),mask2)));
      r7 = Or(r7, And(LShift(r6, 33), And(Not(pos),mask2)));
      r8 = Or(r8, And(LShift(r7, 33), And(Not(pos),mask2)));
      r9 = Or(r9, And(LShift(r8, 33), And(Not(pos),mask2)));
      r10 = Or(r10, And(LShift(r9, 33), And(Not(pos),mask2)));
      r11 = Or(r11, And(LShift(r10, 33), And(Not(pos),mask2)));
      r12 = Or(r12, And(LShift(r11, 33), And(Not(pos),mask2)));
      r13 = Or(r13, And(LShift(r12, 33), And(Not(pos),mask2)));
      r14 = Or(r14, And(LShift(r13, 33), And(Not(pos),mask2)));
      r15 = Or(r15, And(LShift(r14, 33), And(Not(pos),mask2)));
      r16 = Or(r16, And(LShift(r15, 33), And(Not(pos),mask2)));
      r17 = Or(r17, And(LShift(r16, 33), And(Not(pos),mask2)));
      r18 = Or(r18, And(LShift(r17, 33), And(Not(pos),mask2)));
      r19 = Or(r19, And(LShift(r18, 33), And(Not(pos),mask2)));
      r20 = Or(r20, And(LShift(r19, 33), And(Not(pos),mask2)));
      r21 = Or(r21, And(LShift(r20, 33), And(Not(pos),mask2)));
      r22 = Or(r22, And(LShift(r21, 33), And(Not(pos),mask2)));
      r23 = Or(r23, And(LShift(r22, 33), And(Not(pos),mask2)));
      r24 = Or(r24, And(LShift(r23, 33), And(Not(pos),mask2)));
      r25 = Or(r25, And(LShift(r24, 33), And(Not(pos),mask2)));
      r26 = Or(r26, And(LShift(r25, 33), And(Not(pos),mask2)));
      r27 = Or(r27, And(LShift(r26, 33), And(Not(pos),mask2)));
      r28 = Or(r28, And(LShift(r27, 33), And(Not(pos),mask2)));
      r29 = Or(r29, And(LShift(r28, 33), And(Not(pos),mask2)));
      r30 = Or(r30, And(LShift(r29, 33), And(Not(pos),mask2)));
      r31 = Or(r31, And(LShift(r30, 33), And(Not(pos),mask2)));
      r32 = Or(r32, And(LShift(r31, 33), And(Not(pos),mask2)));

      l1 = Or(l1, And(RShift(queens,33), And(Not(pos),mask2)));
      l2 = Or(l2, And(RShift(l1,33), And(Not(pos),mask2)));
      l3 = Or(l3, And(RShift(l2,33), And(Not(pos),mask2)));
      l4 = Or(l4, And(RShift(l3,33), And(Not(pos),mask2)));
      l5 = Or(l5, And(RShift(l4,33), And(Not(pos),mask2)));
      l6 = Or(l6, And(RShift(l5,33), And(Not(pos),mask2)));
      l7 = Or(l7, And(RShift(l6,33), And(Not(pos),mask2)));
      l8 = Or(l8, And(RShift(l7,33), And(Not(pos),mask2)));
      l9 = Or(l9, And(RShift(l8,33), And(Not(pos),mask2)));
      l10 = Or(l10, And(RShift(l9,33), And(Not(pos),mask2)));
      l11 = Or(l11, And(RShift(l10,33), And(Not(pos),mask2)));
      l12 = Or(l12, And(RShift(l11,33), And(Not(pos),mask2)));
      l13 = Or(l13, And(RShift(l12,33), And(Not(pos),mask2)));
      l14 = Or(l14, And(RShift(l13,33), And(Not(pos),mask2)));
      l15 = Or(l15, And(RShift(l14,33), And(Not(pos),mask2)));
      l16 = Or(l16, And(RShift(l15,33), And(Not(pos),mask2)));
      l17 = Or(l17, And(RShift(l16,33), And(Not(pos),mask2)));
      l18 = Or(l18, And(RShift(l17,33), And(Not(pos),mask2)));
      l19 = Or(l19, And(RShift(l18,33), And(Not(pos),mask2)));
      l20 = Or(l20, And(RShift(l19,33), And(Not(pos),mask2)));
      l21 = Or(l21, And(RShift(l20,33), And(Not(pos),mask2)));
      l22 = Or(l22, And(RShift(l21,33), And(Not(pos),mask2)));
      l23 = Or(l23, And(RShift(l22,33), And(Not(pos),mask2)));
      l24 = Or(l24, And(RShift(l23,33), And(Not(pos),mask2)));
      l25 = Or(l25, And(RShift(l24,33), And(Not(pos),mask2)));
      l26 = Or(l26, And(RShift(l25,33), And(Not(pos),mask2)));
      l27 = Or(l27, And(RShift(l26,33), And(Not(pos),mask2)));
      l28 = Or(l28, And(RShift(l27,33), And(Not(pos),mask2)));
      l29 = Or(l29, And(RShift(l28,33), And(Not(pos),mask2)));
      l30 = Or(l30, And(RShift(l29,33), And(Not(pos),mask2)));
      l31 = Or(l31, And(RShift(l30,33), And(Not(pos),mask2)));
      l32 = Or(l32, And(RShift(l31,33), And(Not(pos),mask2)));

    }
    
    if(Positive(And(queens, mask1))) {
      r1 = Or(r1, And(LShift(queens, 33), And(Not(pos),mask1)));
      r2 = Or(r2, And(LShift(r1, 33), And(Not(pos),mask1)));
      r3 = Or(r3, And(LShift(r2, 33), And(Not(pos),mask1)));
      r4 = Or(r4, And(LShift(r3, 33), And(Not(pos),mask1)));
      r5 = Or(r5, And(LShift(r4, 33), And(Not(pos),mask1)));
      r6 = Or(r6, And(LShift(r5, 33), And(Not(pos),mask1)));
      r7 = Or(r7, And(LShift(r6, 33), And(Not(pos),mask1)));
      r8 = Or(r8, And(LShift(r7, 33), And(Not(pos),mask1)));
      r9 = Or(r9, And(LShift(r8, 33), And(Not(pos),mask1)));
      r10 = Or(r10, And(LShift(r9, 33), And(Not(pos),mask1)));
      r11 = Or(r11, And(LShift(r10, 33), And(Not(pos),mask1)));
      r12 = Or(r12, And(LShift(r11, 33), And(Not(pos),mask1)));
      r13 = Or(r13, And(LShift(r12, 33), And(Not(pos),mask1)));
      r14 = Or(r14, And(LShift(r13, 33), And(Not(pos),mask1)));
      r15 = Or(r15, And(LShift(r14, 33), And(Not(pos),mask1)));
      r16 = Or(r16, And(LShift(r15, 33), And(Not(pos),mask1)));
      r17 = Or(r17, And(LShift(r16, 33), And(Not(pos),mask1)));
      r18 = Or(r18, And(LShift(r17, 33), And(Not(pos),mask1)));
      r19 = Or(r19, And(LShift(r18, 33), And(Not(pos),mask1)));
      r20 = Or(r20, And(LShift(r19, 33), And(Not(pos),mask1)));
      r21 = Or(r21, And(LShift(r20, 33), And(Not(pos),mask1)));
      r22 = Or(r22, And(LShift(r21, 33), And(Not(pos),mask1)));
      r23 = Or(r23, And(LShift(r22, 33), And(Not(pos),mask1)));
      r24 = Or(r24, And(LShift(r23, 33), And(Not(pos),mask1)));
      r25 = Or(r25, And(LShift(r24, 33), And(Not(pos),mask1)));
      r26 = Or(r26, And(LShift(r25, 33), And(Not(pos),mask1)));
      r27 = Or(r27, And(LShift(r26, 33), And(Not(pos),mask1)));
      r28 = Or(r28, And(LShift(r27, 33), And(Not(pos),mask1)));
      r29 = Or(r29, And(LShift(r28, 33), And(Not(pos),mask1)));
      r30 = Or(r30, And(LShift(r29, 33), And(Not(pos),mask1)));
      r31 = Or(r31, And(LShift(r30, 33), And(Not(pos),mask1)));
      r32 = Or(r32, And(LShift(r31, 33), And(Not(pos),mask1)));
  

      l1 = Or(l1, And(RShift(queens,33), And(Not(pos),mask1)));
      l2 = Or(l2, And(RShift(l1,33), And(Not(pos),mask1)));
      l3 = Or(l3, And(RShift(l2,33), And(Not(pos),mask1)));
      l4 = Or(l4, And(RShift(l3,33), And(Not(pos),mask1)));
      l5 = Or(l5, And(RShift(l4,33), And(Not(pos),mask1)));
      l6 = Or(l6, And(RShift(l5,33), And(Not(pos),mask1)));
      l7 = Or(l7, And(RShift(l6,33), And(Not(pos),mask1)));
      l8 = Or(l8, And(RShift(l7,33), And(Not(pos),mask1)));
      l9 = Or(l9, And(RShift(l8,33), And(Not(pos),mask1)));
      l10 = Or(l10, And(RShift(l9,33), And(Not(pos),mask1)));
      l11 = Or(l11, And(RShift(l10,33), And(Not(pos),mask1)));
      l12 = Or(l12, And(RShift(l11,33), And(Not(pos),mask1)));
      l13 = Or(l13, And(RShift(l12,33), And(Not(pos),mask1)));
      l14 = Or(l14, And(RShift(l13,33), And(Not(pos),mask1)));
      l15 = Or(l15, And(RShift(l14,33), And(Not(pos),mask1)));
      l16 = Or(l16, And(RShift(l15,33), And(Not(pos),mask1)));
      l17 = Or(l17, And(RShift(l16,33), And(Not(pos),mask1)));
      l18 = Or(l18, And(RShift(l17,33), And(Not(pos),mask1)));
      l19 = Or(l19, And(RShift(l18,33), And(Not(pos),mask1)));
      l20 = Or(l20, And(RShift(l19,33), And(Not(pos),mask1)));
      l21 = Or(l21, And(RShift(l20,33), And(Not(pos),mask1)));
      l22 = Or(l22, And(RShift(l21,33), And(Not(pos),mask1)));
      l23 = Or(l23, And(RShift(l22,33), And(Not(pos),mask1)));
      l24 = Or(l24, And(RShift(l23,33), And(Not(pos),mask1)));
      l25 = Or(l25, And(RShift(l24,33), And(Not(pos),mask1)));
      l26 = Or(l26, And(RShift(l25,33), And(Not(pos),mask1)));
      l27 = Or(l27, And(RShift(l26,33), And(Not(pos),mask1)));
      l28 = Or(l28, And(RShift(l27,33), And(Not(pos),mask1)));
      l29 = Or(l29, And(RShift(l28,33), And(Not(pos),mask1)));
      l30 = Or(l30, And(RShift(l29,33), And(Not(pos),mask1)));
      l31 = Or(l31, And(RShift(l30,33), And(Not(pos),mask1)));
      l32 = Or(l32, And(RShift(l31,33), And(Not(pos),mask1)));
    }
  } 
  
  for(MBOARD32 mask1 = BISHOP2, mask2 = BISHOP2; Positive(mask2); mask1 = RShiftBishop1(mask1,1), mask2 = LShiftBishop1(mask2,1) ) {
     if(Positive(And(queens, mask1))) {
      u1 = Or(u1, And(LShift(queens,31), And(Not(pos),mask1)));
      u2 = Or(u2, And(LShift(u1,31), And(Not(pos),mask1)));
      u3 = Or(u3, And(LShift(u2,31), And(Not(pos),mask1)));
      u4 = Or(u4, And(LShift(u3,31), And(Not(pos),mask1)));
      u5 = Or(u5, And(LShift(u4,31), And(Not(pos),mask1)));
      u6 = Or(u6, And(LShift(u5,31), And(Not(pos),mask1)));
      u7 = Or(u7, And(LShift(u6,31), And(Not(pos),mask1)));
      u8 = Or(u8, And(LShift(u7,31), And(Not(pos),mask1)));
      u9 = Or(u9, And(LShift(u8,31), And(Not(pos),mask1)));
      u10 = Or(u10, And(LShift(u9,31), And(Not(pos),mask1)));
      u11 = Or(u11, And(LShift(u10,31), And(Not(pos),mask1)));
      u12 = Or(u12, And(LShift(u11,31), And(Not(pos),mask1)));
      u13 = Or(u13, And(LShift(u12,31), And(Not(pos),mask1)));
      u14 = Or(u14, And(LShift(u13,31), And(Not(pos),mask1)));
      u15 = Or(u15, And(LShift(u14,31), And(Not(pos),mask1)));
      u16 = Or(u16, And(LShift(u15,31), And(Not(pos),mask1)));
      u17 = Or(u17, And(LShift(u16,31), And(Not(pos),mask1)));
      u18 = Or(u18, And(LShift(u17,31), And(Not(pos),mask1)));
      u19 = Or(u19, And(LShift(u18,31), And(Not(pos),mask1)));
      u20 = Or(u20, And(LShift(u19,31), And(Not(pos),mask1)));
      u21 = Or(u21, And(LShift(u20,31), And(Not(pos),mask1)));
      u22 = Or(u22, And(LShift(u21,31), And(Not(pos),mask1)));
      u23 = Or(u23, And(LShift(u22,31), And(Not(pos),mask1)));
      u24 = Or(u24, And(LShift(u23,31), And(Not(pos),mask1)));
      u25 = Or(u25, And(LShift(u24,31), And(Not(pos),mask1)));
      u26 = Or(u26, And(LShift(u25,31), And(Not(pos),mask1)));
      u27 = Or(u27, And(LShift(u26,31), And(Not(pos),mask1)));
      u28 = Or(u28, And(LShift(u27,31), And(Not(pos),mask1)));
      u29 = Or(u29, And(LShift(u28,31), And(Not(pos),mask1)));
      u30 = Or(u30, And(LShift(u29,31), And(Not(pos),mask1)));
      u31 = Or(u31, And(LShift(u30,31), And(Not(pos),mask1)));
      u32 = Or(u32, And(LShift(u31,31), And(Not(pos),mask1)));

      
      d1 = Or(d1, And(RShift(queens,31), And(Not(pos),mask1)));
      d2 = Or(d2, And(RShift(d1,31), And(Not(pos),mask1)));
      d3 = Or(d3, And(RShift(d2,31), And(Not(pos),mask1)));
      d4 = Or(d4, And(RShift(d3,31), And(Not(pos),mask1)));
      d5 = Or(d5, And(RShift(d4,31), And(Not(pos),mask1)));
      d6 = Or(d6, And(RShift(d5,31), And(Not(pos),mask1)));
      d7 = Or(d7, And(RShift(d6,31), And(Not(pos),mask1)));
      d8 = Or(d8, And(RShift(d7,31), And(Not(pos),mask1)));
      d9 = Or(d9, And(RShift(d8,31), And(Not(pos),mask1)));
      d10 = Or(d10, And(RShift(d9,31), And(Not(pos),mask1)));
      d11 = Or(d11, And(RShift(d10,31), And(Not(pos),mask1)));
      d12 = Or(d12, And(RShift(d11,31), And(Not(pos),mask1)));
      d13 = Or(d13, And(RShift(d12,31), And(Not(pos),mask1)));
      d14 = Or(d14, And(RShift(d13,31), And(Not(pos),mask1)));
      d15 = Or(d15, And(RShift(d14,31), And(Not(pos),mask1)));
      d16 = Or(d16, And(RShift(d15,31), And(Not(pos),mask1)));
      d17 = Or(d17, And(RShift(d16,31), And(Not(pos),mask1)));
      d18 = Or(d18, And(RShift(d17,31), And(Not(pos),mask1)));
      d19 = Or(d19, And(RShift(d18,31), And(Not(pos),mask1)));
      d20 = Or(d20, And(RShift(d19,31), And(Not(pos),mask1)));
      d21 = Or(d21, And(RShift(d20,31), And(Not(pos),mask1)));
      d22 = Or(d22, And(RShift(d21,31), And(Not(pos),mask1)));
      d23 = Or(d23, And(RShift(d22,31), And(Not(pos),mask1)));
      d24 = Or(d24, And(RShift(d23,31), And(Not(pos),mask1)));
      d25 = Or(d25, And(RShift(d24,31), And(Not(pos),mask1)));
      d26 = Or(d26, And(RShift(d25,31), And(Not(pos),mask1)));
      d27 = Or(d27, And(RShift(d26,31), And(Not(pos),mask1)));
      d28 = Or(d28, And(RShift(d27,31), And(Not(pos),mask1)));
      d29 = Or(d29, And(RShift(d28,31), And(Not(pos),mask1)));
      d30 = Or(d30, And(RShift(d29,31), And(Not(pos),mask1)));
      d31 = Or(d31, And(RShift(d30,31), And(Not(pos),mask1)));
      d32 = Or(d32, And(RShift(d31,31), And(Not(pos),mask1)));
    }
    if(Positive(And(queens,mask2))) {
      u1 = Or(u1, And(LShift(queens,31), And(Not(pos),mask2)));
      u2 = Or(u2, And(LShift(u1,31), And(Not(pos),mask2)));
      u3 = Or(u3, And(LShift(u2,31), And(Not(pos),mask2)));
      u4 = Or(u4, And(LShift(u3,31), And(Not(pos),mask2)));
      u5 = Or(u5, And(LShift(u4,31), And(Not(pos),mask2)));
      u6 = Or(u6, And(LShift(u5,31), And(Not(pos),mask2)));
      u7 = Or(u7, And(LShift(u6,31), And(Not(pos),mask2)));
      u8 = Or(u8, And(LShift(u7,31), And(Not(pos),mask2)));
      u9 = Or(u9, And(LShift(u8,31), And(Not(pos),mask2)));
      u10 = Or(u10, And(LShift(u9,31), And(Not(pos),mask2)));
      u11 = Or(u11, And(LShift(u10,31), And(Not(pos),mask2)));
      u12 = Or(u12, And(LShift(u11,31), And(Not(pos),mask2)));
      u13 = Or(u13, And(LShift(u12,31), And(Not(pos),mask2)));
      u14 = Or(u14, And(LShift(u13,31), And(Not(pos),mask2)));
      u15 = Or(u15, And(LShift(u14,31), And(Not(pos),mask2)));
      u16 = Or(u16, And(LShift(u15,31), And(Not(pos),mask2)));
      u17 = Or(u17, And(LShift(u16,31), And(Not(pos),mask2)));
      u18 = Or(u18, And(LShift(u17,31), And(Not(pos),mask2)));
      u19 = Or(u19, And(LShift(u18,31), And(Not(pos),mask2)));
      u20 = Or(u20, And(LShift(u19,31), And(Not(pos),mask2)));
      u21 = Or(u21, And(LShift(u20,31), And(Not(pos),mask2)));
      u22 = Or(u22, And(LShift(u21,31), And(Not(pos),mask2)));
      u23 = Or(u23, And(LShift(u22,31), And(Not(pos),mask2)));
      u24 = Or(u24, And(LShift(u23,31), And(Not(pos),mask2)));
      u25 = Or(u25, And(LShift(u24,31), And(Not(pos),mask2)));
      u26 = Or(u26, And(LShift(u25,31), And(Not(pos),mask2)));
      u27 = Or(u27, And(LShift(u26,31), And(Not(pos),mask2)));
      u28 = Or(u28, And(LShift(u27,31), And(Not(pos),mask2)));
      u29 = Or(u29, And(LShift(u28,31), And(Not(pos),mask2)));
      u30 = Or(u30, And(LShift(u29,31), And(Not(pos),mask2)));
      u31 = Or(u31, And(LShift(u30,31), And(Not(pos),mask2)));
      u32 = Or(u32, And(LShift(u31,31), And(Not(pos),mask2)));

      
      d1 = Or(d1, And(RShift(queens,31), And(Not(pos),mask2)));
      d2 = Or(d2, And(RShift(d1,31), And(Not(pos),mask2)));
      d3 = Or(d3, And(RShift(d2,31), And(Not(pos),mask2)));
      d4 = Or(d4, And(RShift(d3,31), And(Not(pos),mask2)));
      d5 = Or(d5, And(RShift(d4,31), And(Not(pos),mask2)));
      d6 = Or(d6, And(RShift(d5,31), And(Not(pos),mask2)));
      d7 = Or(d7, And(RShift(d6,31), And(Not(pos),mask2)));
      d8 = Or(d8, And(RShift(d7,31), And(Not(pos),mask2)));
      d9 = Or(d9, And(RShift(d8,31), And(Not(pos),mask2)));
      d10 = Or(d10, And(RShift(d9,31), And(Not(pos),mask2)));
      d11 = Or(d11, And(RShift(d10,31), And(Not(pos),mask2)));
      d12 = Or(d12, And(RShift(d11,31), And(Not(pos),mask2)));
      d13 = Or(d13, And(RShift(d12,31), And(Not(pos),mask2)));
      d14 = Or(d14, And(RShift(d13,31), And(Not(pos),mask2)));
      d15 = Or(d15, And(RShift(d14,31), And(Not(pos),mask2)));
      d16 = Or(d16, And(RShift(d15,31), And(Not(pos),mask2)));
      d17 = Or(d17, And(RShift(d16,31), And(Not(pos),mask2)));
      d18 = Or(d18, And(RShift(d17,31), And(Not(pos),mask2)));
      d19 = Or(d19, And(RShift(d18,31), And(Not(pos),mask2)));
      d20 = Or(d20, And(RShift(d19,31), And(Not(pos),mask2)));
      d21 = Or(d21, And(RShift(d20,31), And(Not(pos),mask2)));
      d22 = Or(d22, And(RShift(d21,31), And(Not(pos),mask2)));
      d23 = Or(d23, And(RShift(d22,31), And(Not(pos),mask2)));
      d24 = Or(d24, And(RShift(d23,31), And(Not(pos),mask2)));
      d25 = Or(d25, And(RShift(d24,31), And(Not(pos),mask2)));
      d26 = Or(d26, And(RShift(d25,31), And(Not(pos),mask2)));
      d27 = Or(d27, And(RShift(d26,31), And(Not(pos),mask2)));
      d28 = Or(d28, And(RShift(d27,31), And(Not(pos),mask2)));
      d29 = Or(d29, And(RShift(d28,31), And(Not(pos),mask2)));
      d30 = Or(d30, And(RShift(d29,31), And(Not(pos),mask2)));
      d31 = Or(d31, And(RShift(d30,31), And(Not(pos),mask2)));
      d32 = Or(d32, And(RShift(d31,31), And(Not(pos),mask2)));
      }   
  }
  MBOARD32 bishopMask1 =  Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(r1, r2),r3),r4),r5),r6),r7),r8),l1),
											     l2),l3),l4),l5), l6), l7), l8),
									u1), u2), u3), u4), u5), u6), u7), u8),d1), d2), d3), d4), d5), d6), d7), d8);

  MBOARD32 bishopMask2 =  Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(r9, r10),r11),r12),r13),r14),r15),r16),l9),
											     l10),l11),l12),l13), l14), l15), l16),
									u9), u10), u11), u12), u13), u14), u15), u16),d9), d10), d11), d12), d13), d14), d15), d16);

  MBOARD32 bishopMask3 =  Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(r17, r18),r19),r20),r21),r22),r23),r24),l17),
											     l18),l19),l20),l21), l22), l23), l24),
									u17), u18), u19), u20), u21), u22), u23), u24),d17), d18), d19), d20), d21), d22), d23), d24);

  MBOARD32 bishopMask4 =  Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(r25, r26),r27),r28),r29),r30),r31),r32),l25),
											     l26),l27),l28),l29), l30), l31), l32),
									u25), u26), u27), u28), u29), u30), u31), u32),d25), d26), d27), d28), d29), d30), d31), d32);

  return Or(Or(bishopMask1, bishopMask2),Or(bishopMask3, bishopMask4));
  } 

__device__ __host__ MBOARD32 getQueenMask(MBOARD32 queens) {
  return Or(getRookMask(queens), getBishopMask(queens));
}

__host__ int countWhiteQueensH(MBOARD32 mb) {
 return  __builtin_popcountll(mb.board[0]) + __builtin_popcountll(mb.board[1]) + __builtin_popcountll(mb.board[2]) + __builtin_popcountll(mb.board[3])
   +  __builtin_popcountll(mb.board[4]) + __builtin_popcountll(mb.board[5]) + __builtin_popcountll(mb.board[6]) + __builtin_popcountll(mb.board[7]) +
     __builtin_popcountll(mb.board[8]) + __builtin_popcountll(mb.board[9]) + __builtin_popcountll(mb.board[10]) + __builtin_popcountll(mb.board[11])
   +  __builtin_popcountll(mb.board[12]) + __builtin_popcountll(mb.board[13]) + __builtin_popcountll(mb.board[14]) + __builtin_popcountll(mb.board[15]);
}

__device__ int countWhiteQueensD(MBOARD32 mb) {
  return __popcll(mb.board[0]) + __popcll(mb.board[1]) + __popcll(mb.board[2]) + __popcll(mb.board[3]) + __popcll(mb.board[4]) + __popcll(mb.board[5]) + __popcll(mb.board[6]) + __popcll(mb.board[7]) + __popcll(mb.board[8]) + __popcll(mb.board[9]) + __popcll(mb.board[10]) + __popcll(mb.board[11]) + __popcll(mb.board[12]) + __popcll(mb.board[13]) + __popcll(mb.board[14]) + __popcll(mb.board[15]);
}

__device__ int countBlackQueensD(MBOARD32 mb) {
  MBOARD32 black = Not(getQueenMask(mb));
  return __popcll(black.board[0]) + __popcll(black.board[1]) + __popcll(black.board[2]) + __popcll(black.board[3]) + __popcll(black.board[4]) + __popcll(black.board[5]) + __popcll(black.board[6]) + __popcll(black.board[7]) + __popcll(black.board[8]) + __popcll(black.board[9]) + __popcll(black.board[10]) + __popcll(black.board[11]) + __popcll(black.board[12]) + __popcll(black.board[13]) + __popcll(black.board[14]) + __popcll(black.board[15]);
}
__host__ int countBlackQueensH(MBOARD32 mb) {
  MBOARD32 black = Not(getQueenMask(mb));
  return __builtin_popcountll(black.board[0]) + __builtin_popcountll(black.board[1]) + __builtin_popcountll(black.board[2]) + __builtin_popcountll(black.board[3]) +
    __builtin_popcountll(black.board[4]) + __builtin_popcountll(black.board[5]) + __builtin_popcountll(black.board[6]) + __builtin_popcountll(black.board[7]) +
    __builtin_popcountll(black.board[8]) + __builtin_popcountll(black.board[9]) + __builtin_popcountll(black.board[10]) + __builtin_popcountll(black.board[11]) +
    __builtin_popcountll(black.board[12]) + __builtin_popcountll(black.board[13]) + __builtin_popcountll(black.board[14]) + __builtin_popcountll(black.board[15]);
}

__device__  MBOARD32 findSwap(MBOARD32 queens, int *mx) {
  MBOARD32 qmax = queens;
  int num = 0;
  for(MBOARD32 mask = rookRowMask(); Positive(mask); mask = LShiftRook(mask,1)) {
    MBOARD32 swapped = And(queens, Not(And(mask,queens)));
    int WhiteQ = countWhiteQueensD(swapped);
    int BlackQ = countBlackQueensD(swapped);
    int min = WhiteQ > BlackQ ? BlackQ : WhiteQ;
    if(min > num){
      num = min;
      qmax = swapped;
    } 
  } 
  for(MBOARD32 mask = rookColMask(); Positive(mask); mask = LShift(mask,1)) {
    MBOARD32 swapped = And(queens, Not(And(mask, queens)));
    int WhiteQ = countWhiteQueensD(swapped);
    int BlackQ = countBlackQueensD(swapped);
    int min = WhiteQ > BlackQ ? BlackQ : WhiteQ;
    if(min > num){
      num = min;
      qmax = swapped;
    }
  }
  MBOARD32 BISHOP1 = bishopDiagonal1(); //0x8040201008040201;
  MBOARD32 BISHOP2 = bishopDiagonal2(); //0x0102040810204080;  

  for(MBOARD32 mask1 = BISHOP1, mask2 = BISHOP1; Positive(mask2); mask1 = RShiftBishop1(mask1,1), mask2 = LShiftBishop1(mask2,1)) {
    MBOARD32 swapped = And(queens, Not(And(mask1,queens)));
    int WhiteQ = countWhiteQueensD(swapped);
    int BlackQ = countBlackQueensD(swapped);
    int min = WhiteQ > BlackQ ? BlackQ : WhiteQ;
    if(min > num){
      num = min;
      qmax = swapped;
    }
    
    swapped = And(queens, Not(And(mask2,queens)));
    WhiteQ = countWhiteQueensD(swapped);
    BlackQ = countBlackQueensD(swapped);
    min = WhiteQ > BlackQ ? BlackQ : WhiteQ;
    if(min > num){
      num = min;
      qmax = swapped;
    }
  }

  for(MBOARD32 mask1 = BISHOP2, mask2 = BISHOP2; Positive(mask2); mask1 = RShiftBishop1(mask1,1), mask2 = LShiftBishop1(mask2,1)) {
    MBOARD32 swapped = And(queens, Not(And(mask1, queens)));
    int WhiteQ = countWhiteQueensD(swapped);
    int BlackQ = countBlackQueensD(swapped);
    int min = WhiteQ > BlackQ ? BlackQ : WhiteQ;
    if(min > num){
      num = min;
      qmax = swapped;
    }
    
    swapped = And(queens, Not(And(mask2,queens)));
    WhiteQ = countWhiteQueensD(swapped);
    BlackQ = countBlackQueensD(swapped);
    min = WhiteQ > BlackQ ? BlackQ : WhiteQ;
    if(min > num){
      num = min;
      qmax = swapped;
    }
  }
  *mx = num;
  return qmax;	
} 

__device__  MBOARD32 findSwap2(MBOARD32 queens, int *mx) {
  MBOARD32 qmax = queens;
  int num = 0;
  for(MBOARD32 mask = rookRowMask(); Positive(mask); mask = LShiftRook(mask,1)) {
    MBOARD32 swapped = And(queens, Not(And(mask,queens)));
    int WhiteQ = countWhiteQueensD(swapped);
    int BlackQ = countBlackQueensD(swapped);
    int ms = WhiteQ + BlackQ;
    if(ms > num){
      num = ms;
      qmax = swapped;
    } 
  } 
  for(MBOARD32 mask = rookColMask(); Positive(mask); mask = LShift(mask,1)) {
    MBOARD32 swapped = And(queens, Not(And(mask, queens)));
    int WhiteQ = countWhiteQueensD(swapped);
    int BlackQ = countBlackQueensD(swapped);
    int ms = WhiteQ + BlackQ; 
    if(ms > num){
      num = ms;
      qmax = swapped;
    }
  }
  MBOARD32 BISHOP1 = bishopDiagonal1(); //0x8040201008040201;
  MBOARD32 BISHOP2 = bishopDiagonal2(); //0x0102040810204080;  

  for(MBOARD32 mask1 = BISHOP1, mask2 = BISHOP1; Positive(mask2); mask1 = RShiftBishop1(mask1,1), mask2 = LShiftBishop1(mask2,1)) {
    MBOARD32 swapped = And(queens, Not(And(mask1,queens)));
    int WhiteQ = countWhiteQueensD(swapped);
    int BlackQ = countBlackQueensD(swapped);
    int ms = WhiteQ + BlackQ; 
    if(ms > num){
      num = ms;
      qmax = swapped;
    }
    
    swapped = And(queens, Not(And(mask2,queens)));
    WhiteQ = countWhiteQueensD(swapped);
    BlackQ = countBlackQueensD(swapped);
    ms = WhiteQ + BlackQ; 
    if(ms > num){
      num = ms;
      qmax = swapped;
    }
  }

  for(MBOARD32 mask1 = BISHOP2, mask2 = BISHOP2; Positive(mask2); mask1 = RShiftBishop1(mask1,1), mask2 = LShiftBishop1(mask2,1)) {
    MBOARD32 swapped = And(queens, Not(And(mask1, queens)));
    int WhiteQ = countWhiteQueensD(swapped);
    int BlackQ = countBlackQueensD(swapped);
    int ms = WhiteQ + BlackQ; 
    if(ms > num){
      num = ms;
      qmax = swapped;
    }
    
    swapped = And(queens, Not(And(mask2,queens)));
    WhiteQ = countWhiteQueensD(swapped);
    BlackQ = countBlackQueensD(swapped);
    ms = WhiteQ + BlackQ; 
    if(ms > num){
      num = ms;
      qmax = swapped;
    }
  }
  *mx = num;
  return qmax;	
} 


__global__ void sample(int *mq, MBOARD32 *mxb) {
  //printf("pop %i %i \n", __popcll(0xFFULL), __popcll(~0xFFULL));
  //MBOARD b = {.board = {60753670ULL ,1147788ULL, 34352ULL, 36622ULL}};
  //MBOARD b = genMBOARD(.41,5, 0);
  //  printf("count GPU %i %i \n", countWhiteQueensD(b),countBlackQueensD(b));
  //*mxb = b;
  MBOARD32 mb = {0};
  int c = 0;
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  while(*mq < 150) {
    // mb = genWordNV32((float)id/20000.0,((float)id/1000.0)+1,id);
    //mb = genWordNV32(.08,7,id);
    mb = genWordNV32(.05,5,id);
    //    mb = {.board = {60753670ULL ,1147788ULL, 34352ULL, 36622ULL}};
    int blackQ = countBlackQueensD(mb);
    int whiteQ = countWhiteQueensD(mb);
    //    printf("%i %i %i\n",blackQ, whiteQ, *mq);
   if((whiteQ <= blackQ)) {
    mb = Or(mb, Not(getQueenMask(Not(getQueenMask(mb)))));
    int newBlackQ = countBlackQueensD(mb);
    int newWhiteQ = countWhiteQueensD(mb);
    blackQ = newBlackQ;
    whiteQ = newWhiteQ;
   }
    if(whiteQ >= 130 && blackQ >= 130 ) {
      printf("%i %i %i it %i id %i\n",whiteQ, blackQ, *mq, c, id);
      //printf("%llu %llu %llu %llu\n",mb.board[0],mb.board[1],mb.board[2],mb.board[3]);
      drawBoard(Not(getQueenMask(mb)),mb);
    }
    
    if(*mq > 120 && (whiteQ >= 120 && blackQ >= *mq || whiteQ >= *mq && blackQ >= 120)) {
      int s = 0;
      int sm = 0;
      mb = whiteQ > blackQ ? mb : Not(getQueenMask(mb));
      int mn = whiteQ > blackQ ? blackQ : whiteQ;
      int mx = whiteQ > blackQ ? whiteQ : blackQ;
      MBOARD32 swapped = findSwap2(mb, &sm);
      swapped = findSwap2(swapped, &sm);
      //MBOARD32 swapped = findSwap(mb, &s);
      //if(s < *mq && s > mn && mx > *mq) {
      if(sm > whiteQ + blackQ){
	//mb = whiteQ > blackQ ? mb : Not(getQueenMask(mb));
	swapped = countWhiteQueensD(swapped) > countBlackQueensD(swapped) ? swapped : Not(getQueenMask(swapped));
	swapped = findSwap(swapped, &s);
      }
      if(s > *mq && s > mn){
	whiteQ = s;
	blackQ = s;
	
	printf("%i %i %i it %i id %i\n",whiteQ, blackQ, *mq, c, id);
	  //printf("%llu %llu %llu %llu\n",mb.board[0],mb.board[1],mb.board[2],mb.board[3]);
    
	printf("s = %i\n",s);
	drawBoard(Not(getQueenMask(mb)),mb);
	printf("swapped %i\n",s);
	drawBoard(Not(getQueenMask(swapped)),swapped);      
	
	mb = swapped;
      }
   }
   int mn = blackQ < whiteQ ? blackQ: whiteQ;
   if(mn  > *mq) {
     if(c > 0){
      printf("%i %i %i it %i id %i\n",whiteQ, blackQ, *mq, c, id);
      drawBoard(Not(getQueenMask(mb)),mb);
     }
      //*mq = mn;
     atomicMax(mq,mn);
      *mxb = mb;
    }
    ++c;
  } 
}

/*MBOARD32 sampleH() {
  MBOARD32 mb = {0}, mxb = {0};
  int c = 0;
  int mq = 0;
  while(mq < 38) {
    //  mb = genMBOARDH(.28,6);  36 36
    float p = .2;
    int m = 1;
    //mb = {.board = {(BOARD)rand(),(BOARD)rand(),(BOARD)rand(),(BOARD)rand()}};
    mb = genMBOARDH(p,m);
    // mb = genMBOARDH(.28,5);
    int blackQ = countBlackQueensH(mb);
    int whiteQ = countWhiteQueensH(mb);

    if(whiteQ + blackQ >= 74 && (whiteQ >= 28 && blackQ >= 28)) {
      printf("%i %i %i sum %i p = %f m = %i \n",whiteQ, blackQ,c,mq, p, m);
      drawBoard(Not(getQueenMask(mb)),mb);
    }
    if((whiteQ <= blackQ)) {
      if(whiteQ >= 24){ //28
	printf("max so far %i, %i %i inbalance found p = %f m = %i \n",mq, whiteQ,blackQ,p,m);
	drawBoard(Not(getQueenMask(mb)),mb);
	
	printf("fixing .. \n");
      }
      mb = Or(mb, Not(getQueenMask(Not(getQueenMask(mb)))));
      int newBlackQ = countBlackQueensH(mb);
      int newWhiteQ = countWhiteQueensH(mb);
      if(whiteQ >= 27){
	if(newBlackQ > newWhiteQ) {
	  printf("max so far %i, imbalance remains %i %i p = %f m = %i \n",mq,newWhiteQ, newBlackQ,p,m);
	}
	else printf("max so far %i, imbalance fixed %i %i p = %f m = %i \n", mq,newWhiteQ, newBlackQ,p,m);
	drawBoard(Not(getQueenMask(mb)),mb);
      }
      blackQ = newBlackQ;
      whiteQ = newWhiteQ;
    }
    
    if((whiteQ == blackQ) && whiteQ > mq) {
      mq = whiteQ;
      mxb = mb;
      if(whiteQ > 20){
	printf("%i %i %i hi %i p = %f m = %i \n",whiteQ, blackQ,c,mq, p, m);
	//	printf("%llu %llu %llu %llu \n", mb.board[0], mb.board[1], mb.board[2], mb.board[3]);
	drawBoard(Not(getQueenMask(mb)),mb);
      }
    }
    ++c;
  }
  printf("DONE \n");
  return mxb;
}
*/

int main() {
  int blockSize = 32;
  int blocks = 10000/blockSize;
  MBOARD32 *mxb;
  int * mq;
  hipMallocManaged(&mxb, sizeof(MBOARD32));
  hipMallocManaged(&mq, sizeof(int));
  *mq=0;
  sample<<<20000,blockSize>>>(mq,mxb);
  hipDeviceSynchronize();
  drawBoard(Not(getQueenMask(*mxb)),*mxb); 

  MBOARD32 g = {.board = {0,0,0,1ULL << 22,0,0,0,0}};
  drawBoard(getQueenMask(g),bishopDiagonal1());

  //srand(time(0));
  //sampleH();

  /*
  MBOARD t = {.board = {60753670,1147788, 34352, 36622}};
  printf("count CPU %i %i \n", countWhiteQueensH(t),countBlackQueensH(t));
  //  MBOARD t = {.board = {0,0, 0,  1 << 10 | 1 << 15 }};
    //t = *mxb;
  drawBoard(getQueenMask(t),t);
  drawBoard(Not(getQueenMask(t)),t);
  //drawBoard(bishopDiagonal2(),bishopDiagonal1());
  //for(int i = 0; i < 16; ++i)
  // drawBoard(RShiftBishop1(bishopDiagonal1(),i),t);
  
  printf("%i %i \n", countBlackQueensH(t), countWhiteQueensH(t));
  MBOARD B;
  B.board[0] = 0xFFFFULL;
  B.board[1] = 0xFFFFULL;
  B.board[2] = 0xFFFFULL;
  B.board[3] = 0xFFFFULL;
  drawBoard(B,compliment(B));
  MBOARD C = {.board = {1,1,1,1}};
  drawBoard(C,compliment(C));
  getBit(compliment(B),34);
  printf("get bit %llu \n", (1ULL << 33) & compliment(B).board[0]);
  MBOARD MM = {.board = {1ULL << 60,1,1,1}};
  drawBoard(MM,MM);
  drawBoard(RShift(MM,8),MM);
  drawBoard(bishopDiagonal1(),RShiftBishop1(bishopDiagonal1(),3));
  MBOARD rook = {.board = {0xFFFF,0,0,0}};
  drawBoard(LShiftRook(rook,5),{0});
  drawBoard(getBishopMask(UShiftRook(C,5)),UShiftRook(C,5));
  */  
  
  return 0;
}
