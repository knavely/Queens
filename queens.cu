#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/random.h>
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""

#define N 4
#define R 64 / N
#define M R * R

typedef unsigned long long uint64;
typedef unsigned long long  BOARD;

typedef struct MBOARD {
  BOARD board[N] = {0};
} MBOARD;

BOARD getBit(MBOARD B, BOARD b) {
  BOARD d = b/64;
  BOARD r  = b % 64;
  //  printf("%llu %llu %llu\n", d, r, ((1ULL << r) & B.board[d]));
  return ((1ULL << r) & B.board[d]);
}

MBOARD bishopDiagonal1() {
  BOARD B1 = 1ULL | 1ULL << 17 | 1ULL << 34 | 1ULL << 51;
  BOARD B2 = B1 << 4;
  BOARD B3 = B2 << 4;
  BOARD B4 = B3 << 4;

  return (MBOARD){.board = {B1,B2,B3,B4}};
}

MBOARD bishopDiagonal2() {
  BOARD B1 = 1ULL << 15 | 1ULL << 30 | 1ULL << 45 | 1ULL << 60;
  BOARD B2 = B1 >> 4;
  BOARD B3 = B2 >> 4;
  BOARD B4 = B3 >> 4;

  return (MBOARD){.board = {B1,B2,B3,B4}};
}


MBOARD compliment(MBOARD B) {
  MBOARD C = B;
  for(int i = 0; i < N; ++i) {
    C.board[i] = ~B.board[i];
  }
  return C;
}

MBOARD LShift(MBOARD B,int k) {
  BOARD t = 0; //(B.board[0] & (((1ULL << (k)) - 1) << 64-k));
  for(int i = 0; i < N; ++i) {
    BOARD T = B.board[i];
    B.board[i] = ((B.board[i] << k) | t);
    t = (T & (((1ULL << (k)) - 1) << 64-k)) >> 64-k;	
  }
  return B;
}

MBOARD RShift(MBOARD B,int k) {
  BOARD t = 0; //B.board[0] & (((1ULL << k) - 1) << 64-k);
  for(int i = N-1; i >= 0; --i) {
    BOARD T = B.board[i];
    B.board[i] = ((B.board[i] >> k) | t);
    t = (T & ((1ULL << (k)) - 1)) << 64-k;	
  }
  return B;
}

MBOARD RShiftBishop1(MBOARD mb, int k) {
  MBOARD r = mb;
  for(int i = 0; i < k; ++i) {
    r = RShift(r,1);
    int p = 15; 
    r.board[0] = ~(1ULL << p | 1ULL << (p+16) | 1ULL << (p+32) | 1ULL << (p+48)) & r.board[0];
    r.board[1] = ~(1ULL << p | 1ULL << (p+16) | 1ULL << (p+32) | 1ULL << (p+48)) & r.board[1];
    r.board[2] = ~(1ULL << p | 1ULL << (p+16) | 1ULL << (p+32) | 1ULL << (p+48)) & r.board[2];
    r.board[3] = ~(1ULL << p | 1ULL << (p+16) | 1ULL << (p+32) | 1ULL << (p+48)) & r.board[3];

    /*r.board[1] = ~(1ULL << 15 | 1ULL << 31 | 1ULL << 47 | 1ULL << 63) & r.board[1];
    r.board[2] = ~(1ULL << 15 | 1ULL << 31 | 1ULL << 47 | 1ULL << 63)  & r.board[2];
    r.board[3] = ~(1ULL << 15 | 1ULL << 31 | 1ULL << 47 | 1ULL << 63) & r.board[3]; */
  }
  return r;
}


void drawBoard(MBOARD white, MBOARD black) {
  for(int r = 0; r < R; ++r) {
    for(int c = 0; c < R; ++c) {
      if(getBit(white, r*R + c)) 
	printf(" %s ", "\u2655");
      else if(getBit(black,r*R + c)) 
	printf(" %s ", "\u265B");
      else{  
	printf(" %s ", "\u25A0");
      }
    }
    printf("\n");
  }
  printf("\n");  
}

/*
MBOARD getBishopMask(MBOARD queens) {
  const unsigned long BISHOP1 = 0x8040201008040201;
  const unsigned long BISHOP2 = 0x0102040810204080;
  
  MBOARD bishops = queens;
  MBOARD pos = queens;

  unsigned long r1 = 0,r2 = 0,r3 = 0,r4 = 0,r5 = 0,r6 = 0,r7 = 0,r8 = 0,l1 = 0,l2 = 0,l3= 0,l4 = 0,l5 = 0,
    l6 = 0,l7 = 0,l8 = 0;
  unsigned long u1 = 0,u2 = 0,u3 = 0,u4 = 0,u5 = 0,u6 = 0,u7 = 0,u8 = 0,d1 = 0,d2 = 0,d3 = 0,d4 = 0,d5 = 0
    ,d6 = 0,d7 = 0,d8 = 0;

  
  for(unsigned long mask1 = BISHOP1, mask2 = BISHOP1; mask2 > 0; mask1 = mask1 >> 8, mask2 = mask2 << 8 ) {
    if(queens & mask2) {
      r1 |= (queens << 9) & ~pos & mask2;
      r2 |= (r1 << 9) & ~pos & mask2;
      r3 |= (r2 << 9) & ~pos & mask2;
      r4 |= (r3 << 9) & ~pos & mask2;
      r5 |= (r4 << 9) & ~pos & mask2;
      r6 |= (r5 << 9) & ~pos & mask2;
      r7 |= (r6 << 9) & ~pos & mask2;
      r8 |= (r7 << 9) & ~pos & mask2;

      l1 |= (queens >> 9) & ~pos & mask2;
      l2 |= (l1 >> 9) & ~pos & mask2;
      l3 |= (l2 >> 9) & ~pos & mask2;
      l4 |= (l3 >> 9) & ~pos & mask2;
      l5 |= (l4 >> 9) & ~pos & mask2;
      l6 |= (l5 >> 9) & ~pos & mask2;
      l7 |= (l6 >> 9) & ~pos & mask2;
      l8 |= (l7 >> 9) & ~pos & mask2;
    } 
    if(queens & mask1) {
      r1 |= (queens << 9) & ~pos & mask1;
      r2 |= (r1 <<9)  & ~pos & mask1;
      r3 |= (r2 << 9) & ~pos & mask1;
      r4 |= (r3  << 9) & ~pos & mask1;
      r5 |= (r4 << 9) & ~pos & mask1;
      r6 |= (r5 << 9) & ~pos & mask1;
      r7 |= (r6 << 9) & ~pos & mask1;
      r8 |= (r7  << 9) & ~pos & mask1;

      l1 |= (queens >> 9) & ~pos & mask1;
      l2 |= (l1 >> 9) & ~pos & mask1;
      l3 |= (l2 >> 9) & ~pos & mask1;
      l4 |= (l3 >> 9) & ~pos & mask1;
      l5 |= (l4 >> 9) & ~pos & mask1;
      l6 |= (l5 >> 9) & ~pos & mask1;
      l7 |= (l6 >> 9) & ~pos & mask1;
      l8 |= (l7 >> 9) & ~pos & mask1;
    }  
  } 
  
  for(unsigned long mask1 = BISHOP2, mask2 = BISHOP2; mask2 > 0; mask1 = mask1 >> 8, mask2 = mask2 << 8 ) {
    if(queens & mask1) { 
      u1 |= (queens  << 7) & ~pos & mask1;
      u2 |= (u1 << 7) & ~pos & mask1;
      u3 |= (u2 << 7) & ~pos & mask1;
      u4 |= (u3 << 7) & ~pos & mask1;
      u5 |= (u4 << 7) & ~pos & mask1;
      u6 |= (u5 << 7) & ~pos & mask1;
      u7 |= (u6 << 7) & ~pos & mask1;
      u8 |= (u7 << 7) & ~pos & mask1;

      d1 |= (queens  >> 7) & ~pos & mask1;
      d2 |= (d1 >> 7) & ~pos & mask1;
      d3 |= (d2 >> 7) & ~pos & mask1;
      d4 |= (d3 >> 7) & ~pos & mask1;
      d5 |= (d4 >> 7) & ~pos & mask1;
      d6 |= (d5 >> 7) & ~pos & mask1;
      d7 |= (d6 >> 7) & ~pos & mask1;
      d8 |= (d7 >> 7) & ~pos & mask1;
    }
    if(queens & mask2) {
      u1 |= (queens << 7) & ~pos & mask2;
      u2 |= (u1 << 7) & ~pos & mask2;
      u3 |= (u2 << 7) & ~pos & mask2;
      u4 |= (u3 << 7) & ~pos & mask2;
      u5 |= (u4 << 7) & ~pos & mask2;
      u6 |= (u5 << 7) & ~pos & mask2;
      u7 |= (u6 << 7) & ~pos & mask2;
      u8 |= (u7 << 7) & ~pos & mask2;

      d1 |= (queens >> 7) & ~pos & mask2;
      d2 |= (d1 >> 7) & ~pos & mask2;
      d3 |= (d2 >> 7) & ~pos & mask2;
      d4 |= (d3 >> 7) & ~pos & mask2;
      d5 |= (d4 >> 7) & ~pos & mask2;
      d6 |= (d5 >> 7) & ~pos & mask2;
      d7 |= (d6 >> 7) & ~pos & mask2;
      d8 |= (d7 >> 7) & ~pos & mask2;
    }  
  }
  
  BOARD bishopMask =  r1 | r2 | r3 | r4 | r5 | r6 | r7 | r8 |
    l1 | l2 | l3 | l4 | l5 | l6 | l7 | l8 |
    u1 | u2 | u3 | u4 | u5 | u6 | u7 | u8 |
    d1 | d2 | d3 | d4 | d5 | d6 | d7 | d8;
  
  return bishopMask;
  } */

int main() {
  MBOARD B;
  B.board[0] = 0xFFFFULL;
  B.board[1] = 0xFFFFULL;
  B.board[2] = 0xFFFFULL;
  B.board[3] = 0xFFFFULL;
  drawBoard(B,compliment(B));
  MBOARD C = {.board = {1,1,1,1}};
  drawBoard(C,compliment(C));
  getBit(compliment(B),34);
  printf("get bit %llu \n", (1ULL << 33) & compliment(B).board[0]);
  MBOARD MM = {.board = {1ULL << 60,1,1,1}};
  drawBoard(MM,MM);
  drawBoard(RShift(MM,8),MM);
  drawBoard(bishopDiagonal1(),RShiftBishop1(bishopDiagonal2(),3));
  return 0;
}
