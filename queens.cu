#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/random.h>
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""
//#include "queens.h"
#include "generator.h"
#include "philox.h"
__device__ __host__ BOARD getBit(MBOARD B, BOARD b) {
  BOARD d = b/64;
  BOARD r  = b % 64;
  //  printf("%llu %llu %llu\n", d, r, ((1ULL << r) & B.board[d]));
  return ((1ULL << r) & B.board[d]);
}

__device__ __host__ MBOARD bishopDiagonal1() {
  BOARD B1 = 1ULL | 1ULL << 17 | 1ULL << 34 | 1ULL << 51;
  BOARD B2 = B1 << 4;
  BOARD B3 = B2 << 4;
  BOARD B4 = B3 << 4;

  return (MBOARD){.board = {B1,B2,B3,B4}};
}

__device__ __host__ MBOARD bishopDiagonal2() {
  BOARD B1 = 1ULL << 15 | 1ULL << 30 | 1ULL << 45 | 1ULL << 60;
  BOARD B2 = B1 >> 4;
  BOARD B3 = B2 >> 4;
  BOARD B4 = B3 >> 4;

  return (MBOARD){.board = {B1,B2,B3,B4}};
}

__device__ __host__ MBOARD compliment(MBOARD B) {
  MBOARD C = B;
  for(int i = 0; i < N; ++i) {
    C.board[i] = ~B.board[i];
  }
  return C;
}

__device__ __host__ MBOARD LShift(MBOARD B,int k) {
  BOARD t = 0; //(B.board[0] & (((1ULL << (k)) - 1) << 64-k));
  for(int i = 0; i < N; ++i) {
    BOARD T = B.board[i];
    B.board[i] = ((B.board[i] << k) | t);
    t = (T & (((1ULL << (k)) - 1) << 64-k)) >> 64-k;	
  }
  return B;
}

__device__ __host__ MBOARD RShift(MBOARD B,int k) {
  BOARD t = 0; //B.board[0] & (((1ULL << k) - 1) << 64-k);
  for(int i = N-1; i >= 0; --i) {
    BOARD T = B.board[i];
    B.board[i] = ((B.board[i] >> k) | t);
    t = (T & ((1ULL << (k)) - 1)) << 64-k;	
  }
  return B;
}

__device__ __host__ MBOARD RShiftBishop1(MBOARD mb, int k) {
  MBOARD r = mb;
  for(int i = 0; i < k; ++i) {
    r = RShift(r,1);
    int p = 15; 
    r.board[0] = ~(1ULL << p | 1ULL << (p+16) | 1ULL << (p+32) | 1ULL << (p+48)) & r.board[0];
    r.board[1] = ~(1ULL << p | 1ULL << (p+16) | 1ULL << (p+32) | 1ULL << (p+48)) & r.board[1];
    r.board[2] = ~(1ULL << p | 1ULL << (p+16) | 1ULL << (p+32) | 1ULL << (p+48)) & r.board[2];
    r.board[3] = ~(1ULL << p | 1ULL << (p+16) | 1ULL << (p+32) | 1ULL << (p+48)) & r.board[3];

    /*r.board[1] = ~(1ULL << 15 | 1ULL << 31 | 1ULL << 47 | 1ULL << 63) & r.board[1];
    r.board[2] = ~(1ULL << 15 | 1ULL << 31 | 1ULL << 47 | 1ULL << 63)  & r.board[2];
    r.board[3] = ~(1ULL << 15 | 1ULL << 31 | 1ULL << 47 | 1ULL << 63) & r.board[3]; */
  }
  return r;
}

__device__ __host__ MBOARD LShiftBishop1(MBOARD mb, int k) {
  MBOARD l = mb;
  for(int i = 0; i < k; ++i) {
    l = LShift(l,1);
    int p = 16; 
    l.board[0] = ~(1ULL << p | 1ULL << (p+16) | 1ULL << (p+32) | 1ULL << (p+48)) & l.board[0];
    l.board[1] = ~(1ULL << p | 1ULL << (p+16) | 1ULL << (p+32) | 1ULL << (p+48)) & l.board[1];
    l.board[2] = ~(1ULL << p | 1ULL << (p+16) | 1ULL << (p+32) | 1ULL << (p+48)) & l.board[2];
    l.board[3] = ~(1ULL << p | 1ULL << (p+16) | 1ULL << (p+32) | 1ULL << (p+48)) & l.board[3];
  }
  return l;
}

__device__ __host__ MBOARD LShiftRook(MBOARD mb, int k) {
  MBOARD d = mb;
  for(int i = 0; i < k; ++i) {
    d = LShift(d,16);
  }
  return d;
}

__device__ __host__ MBOARD RShiftRook(MBOARD mb, int k) {
  MBOARD d = mb;
  for(int i = 0; i < k; ++i) {
    d = RShift(d,16);
  }
  return d;
}

__device__ __host__ MBOARD UShiftRook(MBOARD mb, int k) {
  MBOARD d = mb;
  for(int i = 0; i < k; ++i) {
    d = LShift(d,1);
  }
  return d;
}

__device__ __host__ MBOARD DShiftRook(MBOARD mb, int k) {
  MBOARD d = mb;
  for(int i = 0; i < k; ++i) {
    d = RShift(d,1);
  }
  return d;
}

__device__ __host__ void drawBoard(MBOARD white, MBOARD black) {
  for(int r = 0; r < R; ++r) {
    for(int c = 0; c < R; ++c) {
      if(getBit(white, r*R + c)) 
	printf(" %s ", "\u2655");
      else if(getBit(black,r*R + c)) 
	printf(" %s ", "\u265B");
      else{  
	printf(" %s ", "\u25A0");
      }
    }
    printf("\n");
  }
  printf("\n");  
}

__device__ __host__ BOARD Positive(MBOARD B) {
  return (B.board[0] || B.board[1] || B.board[2] || B.board[3]);
  }

__device__ __host__ MBOARD And(MBOARD A, MBOARD B) {
  return (MBOARD){.board = {(A.board[0] & B.board[0]),(A.board[1] & B.board[1]),
      (A.board[2] & B.board[2]), (A.board[3] & B.board[3])}};
}
__device__ __host__ MBOARD Or(MBOARD A, MBOARD B) {
  return (MBOARD){.board = {
      A.board[0] | B.board[0],
      A.board[1] | B.board[1],
      A.board[2] | B.board[2],
      A.board[3] | B.board[3]}};
}

__device__ __host__ MBOARD Not(MBOARD B){
  return (MBOARD){.board = {~(B.board[0]),~(B.board[1]),~(B.board[2]),~(B.board[3])}};
}

__device__ __host__ MBOARD rookRowMask(){
  return {.board = {0xFFFFULL,0ULL,0ULL,0ULL}};
}

__device__ __host__ MBOARD rookColMask(){
  BOARD pattern = 1ULL | (1ULL << 16) | (1ULL << 32) | (1ULL << 48);
  return {.board = {pattern,pattern,pattern,pattern}};

}


__device__ __host__ MBOARD getRookMask(MBOARD queens) {
  MBOARD r1 = {0},r2 = {0},r3 = {0},r4 = {0},r5 = {0},r6 = {0},r7 = {0},r8 = {0},l1 = {0},l2 = {0},l3= {0},l4 = {0},l5 = {0},
    l6 = {0},l7 = {0},l8 = {0};
  MBOARD u1 = {0},u2 = {0},u3 = {0},u4 = {0},u5 = {0},u6 = {0},u7 = {0},u8 = {0},d1 = {0},d2 = {0},d3 = {0},d4 = {0},d5 = {0}
    ,d6 = {0},d7 = {0},d8 = {0};
  MBOARD r9 = {0},r10 = {0},r11 = {0},r12 = {0},r13 = {0},r14 = {0},r15 = {0},r16 = {0},l9 = {0},l10 = {0},l11= {0},l12 = {0},l13 = {0},
    l14 = {0},l15 = {0},l16 = {0};
  MBOARD u9 = {0},u10 = {0},u11 = {0},u12 = {0},u13 = {0},u14 = {0},u15 = {0},u16 = {0},d9 = {0},d10 = {0},d11 = {0},d12 = {0},d13 = {0},d14 = {0},d15 = {0},d16 = {0};

  MBOARD pos = queens;
  for(MBOARD mask = rookRowMask(); Positive(mask); mask = LShiftRook(mask,1)) {
    if(Positive(And(queens, mask))) {
      r1 = Or(r1, And(LShift(queens,1), And(Not(pos),mask)));
      r2 = Or(r2, And(LShift(r1,1), And(Not(pos),mask)));
      r3 = Or(r3, And(LShift(r2,1), And(Not(pos),mask)));
      r4 = Or(r4, And(LShift(r3,1), And(Not(pos),mask)));
      r5 = Or(r5, And(LShift(r4,1), And(Not(pos),mask)));
      r6 = Or(r6, And(LShift(r5,1), And(Not(pos),mask)));
      r7 = Or(r7, And(LShift(r6,1), And(Not(pos),mask)));
      r8 = Or(r8, And(LShift(r7,1), And(Not(pos),mask)));
      r9 = Or(r9, And(LShift(r8,1), And(Not(pos),mask)));
      r10 = Or(r10, And(LShift(r9,1), And(Not(pos),mask)));
      r11 = Or(r11, And(LShift(r10,1), And(Not(pos),mask)));
      r12 = Or(r12, And(LShift(r11,1), And(Not(pos),mask)));
      r13 = Or(r13, And(LShift(r12,1), And(Not(pos),mask)));
      r14 = Or(r14, And(LShift(r13,1), And(Not(pos),mask)));
      r15 = Or(r15, And(LShift(r14,1), And(Not(pos),mask)));
      r16 = Or(r15, And(LShift(r15,1), And(Not(pos),mask)));
      
      l1 = Or(l1, And(RShift(queens,1), And(Not(pos),mask)));
      l2 = Or(l2, And(RShift(l1,1), And(Not(pos),mask)));
      l3 = Or(l3, And(RShift(l2,1), And(Not(pos),mask)));
      l4 = Or(l4, And(RShift(l3,1), And(Not(pos),mask)));
      l5 = Or(l5, And(RShift(l4,1), And(Not(pos),mask)));
      l6 = Or(l6, And(RShift(l5,1), And(Not(pos),mask)));
      l7 = Or(l7, And(RShift(l6,1), And(Not(pos),mask)));
      l8 = Or(l8, And(RShift(l7,1), And(Not(pos),mask)));
      l9 = Or(l9, And(RShift(l8,1), And(Not(pos),mask)));
      l10 = Or(l10, And(RShift(l9,1), And(Not(pos),mask)));
      l11 = Or(l11, And(RShift(l10,1), And(Not(pos),mask)));
      l12 = Or(l12, And(RShift(l11,1), And(Not(pos),mask)));
      l13 = Or(l13, And(RShift(l12,1), And(Not(pos),mask)));
      l14 = Or(l14, And(RShift(l13,1), And(Not(pos),mask)));
      l15 = Or(l15, And(RShift(l14,1), And(Not(pos),mask)));
      l16 = Or(l15, And(RShift(l15,1), And(Not(pos),mask)));    
    }
  } 
  // BOARD pattern = 1ULL | (1ULL << 8) | (1ULL << 16) | (1ULL << 24)
  //| (1ULL << 32ULL) | (1ULL << 40ULL) | (1ULL << 48ULL) | (1ULL << 56ULL);
  for(MBOARD mask = rookColMask(); Positive(mask); mask = LShift(mask,1)) {
      if(Positive(And(queens,mask))) {
      u1 = Or(u1, And(LShift(queens,16), And(Not(pos),mask))); //u1 |= (queens << 8) & ~pos & mask;
      u2 = Or(u2, And(LShift(u1,16), And(Not(pos),mask)));
      u3 = Or(u3, And(LShift(u2,16), And(Not(pos),mask)));
      u4 = Or(u4, And(LShift(u3,16), And(Not(pos),mask)));
      u5 = Or(u5, And(LShift(u4,16), And(Not(pos),mask)));
      u6 = Or(u6, And(LShift(u5,16), And(Not(pos),mask)));
      u7 = Or(u7, And(LShift(u6,16), And(Not(pos),mask)));
      u8 = Or(u8, And(LShift(u7,16), And(Not(pos),mask)));
      u9 = Or(u9, And(LShift(u8,16), And(Not(pos),mask)));
      u10 = Or(u10, And(LShift(u9,16), And(Not(pos),mask)));
      u11 = Or(u11, And(LShift(u10,16), And(Not(pos),mask)));
      u12 = Or(u12, And(LShift(u11,16), And(Not(pos),mask)));
      u13 = Or(u13, And(LShift(u12,16), And(Not(pos),mask)));
      u14 = Or(u14, And(LShift(u13,16), And(Not(pos),mask)));
      u15 = Or(u15, And(LShift(u14,16), And(Not(pos),mask)));
      u16 = Or(u15, And(LShift(u15,16), And(Not(pos),mask)));
      
      d1 = Or(d1, And(RShift(queens,16), And(Not(pos),mask)));
      d2 = Or(d2, And(RShift(d1,16), And(Not(pos),mask)));
      d3 = Or(d3, And(RShift(d2,16), And(Not(pos),mask)));
      d4 = Or(d4, And(RShift(d3,16), And(Not(pos),mask)));
      d5 = Or(d5, And(RShift(d4,16), And(Not(pos),mask)));
      d6 = Or(d6, And(RShift(d5,16), And(Not(pos),mask)));
      d7 = Or(d7, And(RShift(d6,16), And(Not(pos),mask)));
      d8 = Or(d8, And(RShift(d7,16), And(Not(pos),mask)));
      d9 = Or(d9, And(RShift(d8,16), And(Not(pos),mask)));
      d10 = Or(d10, And(RShift(d9,16), And(Not(pos),mask)));
      d11 = Or(d11, And(RShift(d10,16), And(Not(pos),mask)));
      d12 = Or(d12, And(RShift(d11,16), And(Not(pos),mask)));
      d13 = Or(d13, And(RShift(d12,16), And(Not(pos),mask)));
      d14 = Or(d14, And(RShift(d13,16), And(Not(pos),mask)));
      d15 = Or(d15, And(RShift(d14,16), And(Not(pos),mask)));
      d16 = Or(d15, And(RShift(d15,16), And(Not(pos),mask)));
      
    } 
  }
  MBOARD rookMask1 =  Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(r1, r2),r3),r4),r5),r6),r7),r8),l1),
											     l2),l3),l4),l5), l6), l7), l8),
									u1), u2), u3), u4), u5), u6), u7), u8),d1), d2), d3), d4), d5), d6), d7), d8);

  MBOARD rookMask2 =  Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(r9, r10),r11),r12),r13),r14),r15),r16),l9),
											     l10),l11),l12),l13), l14), l15), l16),
									u9), u10), u11), u12), u13), u14), u15), u16),d9), d10), d11), d12), d13), d14), d15), d16);
  
  return Or(queens,Or(rookMask1, rookMask2));	

} 

__device__ __host__ MBOARD getBishopMask(MBOARD queens) {
  MBOARD BISHOP1 = bishopDiagonal1(); //0x8040201008040201;
  MBOARD BISHOP2 = bishopDiagonal2(); //0x0102040810204080;  
  MBOARD pos = queens;

  MBOARD r1 = {0},r2 = {0},r3 = {0},r4 = {0},r5 = {0},r6 = {0},r7 = {0},r8 = {0},l1 = {0},l2 = {0},l3= {0},l4 = {0},l5 = {0},
    l6 = {0},l7 = {0},l8 = {0};
  MBOARD u1 = {0},u2 = {0},u3 = {0},u4 = {0},u5 = {0},u6 = {0},u7 = {0},u8 = {0},d1 = {0},d2 = {0},d3 = {0},d4 = {0},d5 = {0}
    ,d6 = {0},d7 = {0},d8 = {0};
  MBOARD r9 = {0},r10 = {0},r11 = {0},r12 = {0},r13 = {0},r14 = {0},r15 = {0},r16 = {0},l9 = {0},l10 = {0},l11= {0},l12 = {0},l13 = {0},
    l14 = {0},l15 = {0},l16 = {0};
  MBOARD u9 = {0},u10 = {0},u11 = {0},u12 = {0},u13 = {0},u14 = {0},u15 = {0},u16 = {0},d9 = {0},d10 = {0},d11 = {0},d12 = {0},d13 = {0},d14 = {0},d15 = {0},d16 = {0};
  
  for(MBOARD mask1 = BISHOP1, mask2 = BISHOP1; Positive(mask2); mask1 = RShiftBishop1(mask1,1), mask2 = LShiftBishop1(mask2,1)) {
    if(Positive(And(queens,mask2))) {
      //drawBoard(queens,mask2);
      //drawBoard(queens,r5);
      r1 = Or(r1, And(LShift(queens, 17), And(Not(pos),mask2)));
      r2 = Or(r2, And(LShift(r1, 17), And(Not(pos),mask2)));
      r3 = Or(r3, And(LShift(r2, 17), And(Not(pos),mask2)));
      r4 = Or(r4, And(LShift(r3, 17), And(Not(pos),mask2)));
      r5 = Or(r5, And(LShift(r4, 17), And(Not(pos),mask2)));
      r6 = Or(r6, And(LShift(r5, 17), And(Not(pos),mask2)));
      r7 = Or(r7, And(LShift(r6, 17), And(Not(pos),mask2)));
      r8 = Or(r8, And(LShift(r7, 17), And(Not(pos),mask2)));
      r9 = Or(r9, And(LShift(r8, 17), And(Not(pos),mask2)));
      r10 = Or(r10, And(LShift(r9, 17), And(Not(pos),mask2)));
      r11 = Or(r11, And(LShift(r10, 17), And(Not(pos),mask2)));
      r12 = Or(r12, And(LShift(r11, 17), And(Not(pos),mask2)));
      r13 = Or(r13, And(LShift(r12, 17), And(Not(pos),mask2)));
      r14 = Or(r14, And(LShift(r13, 17), And(Not(pos),mask2)));
      r15 = Or(r15, And(LShift(r14, 17), And(Not(pos),mask2)));
      r16 = Or(r16, And(LShift(r15, 17), And(Not(pos),mask2)));
  
      l1 = Or(l1, And(RShift(queens,17), And(Not(pos),mask2)));
      l2 = Or(l2, And(RShift(l1,17), And(Not(pos),mask2)));
      l3 = Or(l3, And(RShift(l2,17), And(Not(pos),mask2)));
      l4 = Or(l4, And(RShift(l3,17), And(Not(pos),mask2)));
      l5 = Or(l5, And(RShift(l4,17), And(Not(pos),mask2)));
      l6 = Or(l6, And(RShift(l5,17), And(Not(pos),mask2)));
      l7 = Or(l7, And(RShift(l6,17), And(Not(pos),mask2)));
      l8 = Or(l8, And(RShift(l7,17), And(Not(pos),mask2)));
      l9 = Or(l9, And(RShift(l8,17), And(Not(pos),mask2)));
      l10 = Or(l10, And(RShift(l9,17), And(Not(pos),mask2)));
      l11 = Or(l11, And(RShift(l10,17), And(Not(pos),mask2)));
      l12 = Or(l12, And(RShift(l11,17), And(Not(pos),mask2)));
      l13 = Or(l13, And(RShift(l12,17), And(Not(pos),mask2)));
      l14 = Or(l14, And(RShift(l13,17), And(Not(pos),mask2)));
      l15 = Or(l15, And(RShift(l14,17), And(Not(pos),mask2)));
      l16 = Or(l16, And(RShift(l15,17), And(Not(pos),mask2)));
    }
    
    if(Positive(And(queens, mask1))) {
      r1 = Or(r1, And(LShift(queens, 17), And(Not(pos),mask1)));
      r2 = Or(r2, And(LShift(r1, 17), And(Not(pos),mask1)));
      r3 = Or(r3, And(LShift(r2, 17), And(Not(pos),mask1)));
      r4 = Or(r4, And(LShift(r3, 17), And(Not(pos),mask1)));
      r5 = Or(r5, And(LShift(r4, 17), And(Not(pos),mask1)));
      r6 = Or(r6, And(LShift(r5, 17), And(Not(pos),mask1)));
      r7 = Or(r7, And(LShift(r6, 17), And(Not(pos),mask1)));
      r8 = Or(r8, And(LShift(r7, 17), And(Not(pos),mask1)));
      r9 = Or(r9, And(LShift(r8, 17), And(Not(pos),mask1)));
      r10 = Or(r10, And(LShift(r9, 17), And(Not(pos),mask1)));
      r11 = Or(r11, And(LShift(r10, 17), And(Not(pos),mask1)));
      r12 = Or(r12, And(LShift(r11, 17), And(Not(pos),mask1)));
      r13 = Or(r13, And(LShift(r12, 17), And(Not(pos),mask1)));
      r14 = Or(r14, And(LShift(r13, 17), And(Not(pos),mask1)));
      r15 = Or(r15, And(LShift(r14, 17), And(Not(pos),mask1)));
      r16 = Or(r16, And(LShift(r15, 17), And(Not(pos),mask1)));
  

      l1 = Or(l1, And(RShift(queens,17), And(Not(pos),mask1)));
      l2 = Or(l2, And(RShift(l1,17), And(Not(pos),mask1)));
      l3 = Or(l3, And(RShift(l2,17), And(Not(pos),mask1)));
      l4 = Or(l4, And(RShift(l3,17), And(Not(pos),mask1)));
      l5 = Or(l5, And(RShift(l4,17), And(Not(pos),mask1)));
      l6 = Or(l6, And(RShift(l5,17), And(Not(pos),mask1)));
      l7 = Or(l7, And(RShift(l6,17), And(Not(pos),mask1)));
      l8 = Or(l8, And(RShift(l7,17), And(Not(pos),mask1)));
      l9 = Or(l9, And(RShift(l8,17), And(Not(pos),mask1)));
      l10 = Or(l10, And(RShift(l9,17), And(Not(pos),mask1)));
      l11 = Or(l11, And(RShift(l10,17), And(Not(pos),mask1)));
      l12 = Or(l12, And(RShift(l11,17), And(Not(pos),mask1)));
      l13 = Or(l13, And(RShift(l12,17), And(Not(pos),mask1)));
      l14 = Or(l14, And(RShift(l13,17), And(Not(pos),mask1)));
      l15 = Or(l15, And(RShift(l14,17), And(Not(pos),mask1)));
      l16 = Or(l16, And(RShift(l15,17), And(Not(pos),mask1)));
    }
  } 
  
  for(MBOARD mask1 = BISHOP2, mask2 = BISHOP2; Positive(mask2); mask1 = RShiftBishop1(mask1,1), mask2 = LShiftBishop1(mask2,1) ) {
     if(Positive(And(queens, mask1))) {
      u1 = Or(u1, And(LShift(queens,15), And(Not(pos),mask1)));
      u2 = Or(u2, And(LShift(u1,15), And(Not(pos),mask1)));
      u3 = Or(u3, And(LShift(u2,15), And(Not(pos),mask1)));
      u4 = Or(u4, And(LShift(u3,15), And(Not(pos),mask1)));
      u5 = Or(u5, And(LShift(u4,15), And(Not(pos),mask1)));
      u6 = Or(u6, And(LShift(u5,15), And(Not(pos),mask1)));
      u7 = Or(u7, And(LShift(u6,15), And(Not(pos),mask1)));
      u8 = Or(u8, And(LShift(u7,15), And(Not(pos),mask1)));
      u9 = Or(u9, And(LShift(u8,15), And(Not(pos),mask1)));
      u10 = Or(u10, And(LShift(u9,15), And(Not(pos),mask1)));
      u11 = Or(u11, And(LShift(u10,15), And(Not(pos),mask1)));
      u12 = Or(u12, And(LShift(u11,15), And(Not(pos),mask1)));
      u13 = Or(u13, And(LShift(u12,15), And(Not(pos),mask1)));
      u14 = Or(u14, And(LShift(u13,15), And(Not(pos),mask1)));
      u15 = Or(u15, And(LShift(u14,15), And(Not(pos),mask1)));
      u16 = Or(u16, And(LShift(u15,15), And(Not(pos),mask1)));

      d1 = Or(d1, And(RShift(queens,15), And(Not(pos),mask1)));
      d2 = Or(d2, And(RShift(d1,15), And(Not(pos),mask1)));
      d3 = Or(d3, And(RShift(d2,15), And(Not(pos),mask1)));
      d4 = Or(d4, And(RShift(d3,15), And(Not(pos),mask1)));
      d5 = Or(d5, And(RShift(d4,15), And(Not(pos),mask1)));
      d6 = Or(d6, And(RShift(d5,15), And(Not(pos),mask1)));
      d7 = Or(d7, And(RShift(d6,15), And(Not(pos),mask1)));
      d8 = Or(d8, And(RShift(d7,15), And(Not(pos),mask1)));
      d9 = Or(d9, And(RShift(d8,15), And(Not(pos),mask1)));
      d10 = Or(d10, And(RShift(d9,15), And(Not(pos),mask1)));
      d11 = Or(d11, And(RShift(d10,15), And(Not(pos),mask1)));
      d12 = Or(d12, And(RShift(d11,15), And(Not(pos),mask1)));
      d13 = Or(d13, And(RShift(d12,15), And(Not(pos),mask1)));
      d14 = Or(d14, And(RShift(d13,15), And(Not(pos),mask1)));
      d15 = Or(d15, And(RShift(d14,15), And(Not(pos),mask1)));
      d16 = Or(d16, And(RShift(d15,15), And(Not(pos),mask1)));
    }
    if(Positive(And(queens,mask2))) {
      u1 = Or(u1, And(LShift(queens,15), And(Not(pos),mask2)));
      u2 = Or(u2, And(LShift(u1,15), And(Not(pos),mask2)));
      u3 = Or(u3, And(LShift(u2,15), And(Not(pos),mask2)));
      u4 = Or(u4, And(LShift(u3,15), And(Not(pos),mask2)));
      u5 = Or(u5, And(LShift(u4,15), And(Not(pos),mask2)));
      u6 = Or(u6, And(LShift(u5,15), And(Not(pos),mask2)));
      u7 = Or(u7, And(LShift(u6,15), And(Not(pos),mask2)));
      u8 = Or(u8, And(LShift(u7,15), And(Not(pos),mask2)));
      u9 = Or(u9, And(LShift(u8,15), And(Not(pos),mask2)));
      u10 = Or(u10, And(LShift(u9,15), And(Not(pos),mask2)));
      u11 = Or(u11, And(LShift(u10,15), And(Not(pos),mask2)));
      u12 = Or(u12, And(LShift(u11,15), And(Not(pos),mask2)));
      u13 = Or(u13, And(LShift(u12,15), And(Not(pos),mask2)));
      u14 = Or(u14, And(LShift(u13,15), And(Not(pos),mask2)));
      u15 = Or(u15, And(LShift(u14,15), And(Not(pos),mask2)));
      u16 = Or(u16, And(LShift(u15,15), And(Not(pos),mask2)));

      d1 = Or(d1, And(RShift(queens,15), And(Not(pos),mask2)));
      d2 = Or(d2, And(RShift(d1,15), And(Not(pos),mask2)));
      d3 = Or(d3, And(RShift(d2,15), And(Not(pos),mask2)));
      d4 = Or(d4, And(RShift(d3,15), And(Not(pos),mask2)));
      d5 = Or(d5, And(RShift(d4,15), And(Not(pos),mask2)));
      d6 = Or(d6, And(RShift(d5,15), And(Not(pos),mask2)));
      d7 = Or(d7, And(RShift(d6,15), And(Not(pos),mask2)));
      d8 = Or(d8, And(RShift(d7,15), And(Not(pos),mask2)));
      d9 = Or(d9, And(RShift(d8,15), And(Not(pos),mask2)));
      d10 = Or(d10, And(RShift(d9,15), And(Not(pos),mask2)));
      d11 = Or(d11, And(RShift(d10,15), And(Not(pos),mask2)));
      d12 = Or(d12, And(RShift(d11,15), And(Not(pos),mask2)));
      d13 = Or(d13, And(RShift(d12,15), And(Not(pos),mask2)));
      d14 = Or(d14, And(RShift(d13,15), And(Not(pos),mask2)));
      d15 = Or(d15, And(RShift(d14,15), And(Not(pos),mask2)));
      d16 = Or(d16, And(RShift(d15,15), And(Not(pos),mask2)));
      }   
  }
  
  MBOARD bishopMask1 =  Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(r1, r2),r3),r4),r5),r6),r7),r8),l1),
											     l2),l3),l4),l5), l6), l7), l8),
									u1), u2), u3), u4), u5), u6), u7), u8),d1), d2), d3), d4), d5), d6), d7), d8);

  MBOARD bishopMask2 =  Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(r9, r10),r11),r12),r13),r14),r15),r16),l9),
											     l10),l11),l12),l13), l14), l15), l16),
									u9), u10), u11), u12), u13), u14), u15), u16),d9), d10), d11), d12), d13), d14), d15), d16);
  
  return Or(bishopMask1, bishopMask2);
  } 

__device__ __host__ MBOARD getQueenMask(MBOARD queens) {
  return Or(getRookMask(queens), getBishopMask(queens));
}

__host__ int countWhiteQueensH(MBOARD mb) {
 return  __builtin_popcountll(mb.board[0]) + __builtin_popcountll(mb.board[1]) + __builtin_popcountll(mb.board[2]) + __builtin_popcountll(mb.board[3]);
}

__device__ int countWhiteQueensD(MBOARD mb) {
  return __popcll(mb.board[0]) + __popcll(mb.board[1]) + __popcll(mb.board[2]) + __popcll(mb.board[3]);
}

__device__ int countBlackQueensD(MBOARD mb) {
  MBOARD black = Not(getQueenMask(mb));
  return __popcll(black.board[0]) + __popcll(black.board[1]) + __popcll(black.board[2]) + __popcll(black.board[3]);
}
__host__ int countBlackQueensH(MBOARD mb) {
  MBOARD black = Not(getQueenMask(mb));
  return __builtin_popcountll(black.board[0]) + __builtin_popcountll(black.board[1]) + __builtin_popcountll(black.board[2]) + __builtin_popcountll(black.board[3]);
}

__global__ void sample(int *mq, MBOARD *mxb) {
  MBOARD mb = {0};
  int i = blockIdx.x*blockDim.x * threadIdx.x;
  int c = 0;
  while(c < 1000) {
    mb = genMBOARD(.41,5, i);
    int blackQ = countBlackQueensD(mb);
    int whiteQ = countWhiteQueensD(mb);
 
    if(whiteQ == blackQ && whiteQ > *mq) {
      atomicMax(mq,whiteQ);
      *mxb = mb;
      if(whiteQ > 10){
	printf("%i %i %i hi %i\n",whiteQ, blackQ,c,*mq);
	printf("%llu %llu %llu %llu \n", mb.board[0], mb.board[1], mb.board[2], mb.board[3]);
	//drawBoard(Not(getQueenMask(mb)),mb);
      }
    }
    ++c;
  }
}

MBOARD sampleH() {
  MBOARD mb = {0}, mxb = {0};
  int c = 0;
  int mq = 0;
  while(c < 100000000) {
    mb = genMBOARDH(.41,7);
    int blackQ = countBlackQueensH(mb);
    int whiteQ = countWhiteQueensH(mb);
 
    if((whiteQ == blackQ) && whiteQ > mq) {
      mq = whiteQ;
      mxb = mb;
      if(whiteQ > 20){
	printf("%i %i %i hi %i\n",whiteQ, blackQ,c,mq);
	//	printf("%llu %llu %llu %llu \n", mb.board[0], mb.board[1], mb.board[2], mb.board[3]);
	drawBoard(Not(getQueenMask(mb)),mb);
      }
    }
    ++c;
  }
  return mxb;
}

int main() {
  int blockSize = 128;
  int blocks = 4000/blockSize;
  MBOARD *mxb;
  int * mq;
  hipMallocManaged(&mxb, sizeof(MBOARD));
  hipMallocManaged(&mq, sizeof(int));
  //sample<<<1,1>>>(mq,mxb);
  sampleH();
  hipDeviceSynchronize();
    MBOARD t = {.board = {60753670,1147788, 34352, 36622}};
  //  MBOARD t = {.board = {0,0, 0,  1 << 10 | 1 << 15 }};
  drawBoard(getQueenMask(t),t);
  drawBoard(t,Not(getQueenMask(t)));
  //drawBoard(bishopDiagonal2(),bishopDiagonal1());
  //for(int i = 0; i < 16; ++i)
  // drawBoard(RShiftBishop1(bishopDiagonal1(),i),t);
  /*
  printf("%i %i \n", countBlackQueensH(t), countWhiteQueensH(t));
  MBOARD B;
  B.board[0] = 0xFFFFULL;
  B.board[1] = 0xFFFFULL;
  B.board[2] = 0xFFFFULL;
  B.board[3] = 0xFFFFULL;
  drawBoard(B,compliment(B));
  MBOARD C = {.board = {1,1,1,1}};
  drawBoard(C,compliment(C));
  getBit(compliment(B),34);
  printf("get bit %llu \n", (1ULL << 33) & compliment(B).board[0]);
  MBOARD MM = {.board = {1ULL << 60,1,1,1}};
  drawBoard(MM,MM);
  drawBoard(RShift(MM,8),MM);
  drawBoard(bishopDiagonal1(),RShiftBishop1(bishopDiagonal1(),3));
  MBOARD rook = {.board = {0xFFFF,0,0,0}};
  drawBoard(LShiftRook(rook,5),{0});
  drawBoard(getBishopMask(UShiftRook(C,5)),UShiftRook(C,5));
  MBOARD g = {.board = {0,0,0,1ULL << 44}};
  drawBoard(getQueenMask(g),{0});
  */
  return 0;
}
