#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/random.h>
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""

#define N 4
#define R 64 / N
#define M R * R

typedef unsigned long long uint64;
typedef unsigned long long  BOARD;

typedef struct MBOARD {
  BOARD board[N] = {0};
} MBOARD;

BOARD getBit(MBOARD B, BOARD b) {
  BOARD d = b/64;
  BOARD r  = b % 64;
  //  printf("%llu %llu %llu\n", d, r, ((1ULL << r) & B.board[d]));
  return ((1ULL << r) & B.board[d]);
}

MBOARD bishopDiagonal1() {
  BOARD B1 = 1ULL | 1ULL << 17 | 1ULL << 34 | 1ULL << 51;
  BOARD B2 = B1 << 4;
  BOARD B3 = B2 << 4;
  BOARD B4 = B3 << 4;

  return (MBOARD){.board = {B1,B2,B3,B4}};
}

MBOARD bishopDiagonal2() {
  BOARD B1 = 1ULL << 15 | 1ULL << 30 | 1ULL << 45 | 1ULL << 60;
  BOARD B2 = B1 >> 4;
  BOARD B3 = B2 >> 4;
  BOARD B4 = B3 >> 4;

  return (MBOARD){.board = {B1,B2,B3,B4}};
}

MBOARD compliment(MBOARD B) {
  MBOARD C = B;
  for(int i = 0; i < N; ++i) {
    C.board[i] = ~B.board[i];
  }
  return C;
}

MBOARD LShift(MBOARD B,int k) {
  BOARD t = 0; //(B.board[0] & (((1ULL << (k)) - 1) << 64-k));
  for(int i = 0; i < N; ++i) {
    BOARD T = B.board[i];
    B.board[i] = ((B.board[i] << k) | t);
    t = (T & (((1ULL << (k)) - 1) << 64-k)) >> 64-k;	
  }
  return B;
}

MBOARD RShift(MBOARD B,int k) {
  BOARD t = 0; //B.board[0] & (((1ULL << k) - 1) << 64-k);
  for(int i = N-1; i >= 0; --i) {
    BOARD T = B.board[i];
    B.board[i] = ((B.board[i] >> k) | t);
    t = (T & ((1ULL << (k)) - 1)) << 64-k;	
  }
  return B;
}

MBOARD RShiftBishop1(MBOARD mb, int k) {
  MBOARD r = mb;
  for(int i = 0; i < k; ++i) {
    r = RShift(r,1);
    int p = 15; 
    r.board[0] = ~(1ULL << p | 1ULL << (p+16) | 1ULL << (p+32) | 1ULL << (p+48)) & r.board[0];
    r.board[1] = ~(1ULL << p | 1ULL << (p+16) | 1ULL << (p+32) | 1ULL << (p+48)) & r.board[1];
    r.board[2] = ~(1ULL << p | 1ULL << (p+16) | 1ULL << (p+32) | 1ULL << (p+48)) & r.board[2];
    r.board[3] = ~(1ULL << p | 1ULL << (p+16) | 1ULL << (p+32) | 1ULL << (p+48)) & r.board[3];

    /*r.board[1] = ~(1ULL << 15 | 1ULL << 31 | 1ULL << 47 | 1ULL << 63) & r.board[1];
    r.board[2] = ~(1ULL << 15 | 1ULL << 31 | 1ULL << 47 | 1ULL << 63)  & r.board[2];
    r.board[3] = ~(1ULL << 15 | 1ULL << 31 | 1ULL << 47 | 1ULL << 63) & r.board[3]; */
  }
  return r;
}

MBOARD LShiftBishop1(MBOARD mb, int k) {
  MBOARD l = mb;
  for(int i = 0; i < k; ++i) {
    l = LShift(l,1);
    int p = 16; 
    l.board[0] = ~(1ULL << p | 1ULL << (p+16) | 1ULL << (p+32) | 1ULL << (p+48)) & l.board[0];
    l.board[1] = ~(1ULL << p | 1ULL << (p+16) | 1ULL << (p+32) | 1ULL << (p+48)) & l.board[1];
    l.board[2] = ~(1ULL << p | 1ULL << (p+16) | 1ULL << (p+32) | 1ULL << (p+48)) & l.board[2];
    l.board[3] = ~(1ULL << p | 1ULL << (p+16) | 1ULL << (p+32) | 1ULL << (p+48)) & l.board[3];
  }
  return l;
}

MBOARD LShiftRook(MBOARD mb, int k) {
  MBOARD d = mb;
  for(int i = 0; i < k; ++i) {
    d = LShift(d,16);
  }
  return d;
}

MBOARD RShiftRook(MBOARD mb, int k) {
  MBOARD d = mb;
  for(int i = 0; i < k; ++i) {
    d = RShift(d,16);
  }
  return d;
}

MBOARD UShiftRook(MBOARD mb, int k) {
  MBOARD d = mb;
  for(int i = 0; i < k; ++i) {
    d = LShift(d,1);
  }
  return d;
}

MBOARD DShiftRook(MBOARD mb, int k) {
  MBOARD d = mb;
  for(int i = 0; i < k; ++i) {
    d = RShift(d,1);
  }
  return d;
}

void drawBoard(MBOARD white, MBOARD black) {
  for(int r = 0; r < R; ++r) {
    for(int c = 0; c < R; ++c) {
      if(getBit(white, r*R + c)) 
	printf(" %s ", "\u2655");
      else if(getBit(black,r*R + c)) 
	printf(" %s ", "\u265B");
      else{  
	printf(" %s ", "\u25A0");
      }
    }
    printf("\n");
  }
  printf("\n");  
}

BOARD Positive(MBOARD B) {
  return (B.board[0] || B.board[1] || B.board[2] || B.board[3]);
  }

MBOARD And(MBOARD A, MBOARD B) {
  return (MBOARD){.board = {(A.board[0] & B.board[0]),(A.board[1] & B.board[1]),
      (A.board[2] & B.board[2]), (A.board[3] & B.board[3])}};
}
MBOARD Or(MBOARD A, MBOARD B) {
  return (MBOARD){.board = {
      A.board[0] | B.board[0],
      A.board[1] | B.board[1],
      A.board[2] | B.board[2],
      A.board[3] | B.board[3]}};
}

MBOARD Not(MBOARD B){
  return (MBOARD){.board = {~(B.board[0]),~(B.board[1]),~(B.board[2]),~(B.board[3])}};
}

MBOARD getBishopMask(MBOARD queens) {
  MBOARD BISHOP1 = bishopDiagonal1(); //0x8040201008040201;
  MBOARD BISHOP2 = bishopDiagonal2(); //0x0102040810204080;
  
  MBOARD pos = queens;

  MBOARD r1 = {0},r2 = {0},r3 = {0},r4 = {0},r5 = {0},r6 = {0},r7 = {0},r8 = {0},l1 = {0},l2 = {0},l3= {0},l4 = {0},l5 = {0},
    l6 = {0},l7 = {0},l8 = {0};
  MBOARD u1 = {0},u2 = {0},u3 = {0},u4 = {0},u5 = {0},u6 = {0},u7 = {0},u8 = {0},d1 = {0},d2 = {0},d3 = {0},d4 = {0},d5 = {0}
    ,d6 = {0},d7 = {0},d8 = {0};
  MBOARD r9 = {0},r10 = {0},r11 = {0},r12 = {0},r13 = {0},r14 = {0},r15 = {0},r16 = {0},l9 = {0},l10 = {0},l11= {0},l12 = {0},l13 = {0},
    l14 = {0},l15 = {0},l16 = {0};
  MBOARD u9 = {0},u10 = {0},u11 = {0},u12 = {0},u13 = {0},u14 = {0},u15 = {0},u16 = {0},d9 = {0},d10 = {0},d11 = {0},d12 = {0},d13 = {0},d14 = {0},d15 = {0},d16 = {0};

  
  for(MBOARD mask1 = BISHOP1, mask2 = BISHOP1; Positive(mask2); mask1 = RShiftBishop1(mask1,1), mask2 = LShiftBishop1(mask2,1)) {
     drawBoard(queens,r1);
    if(Positive(And(queens,mask2))) {
      //drawBoard(queens,mask2);
      //drawBoard(queens,r5);
      r1 = Or(r1, And(LShift(queens, 17), And(Not(pos),mask2)));
      r2 = Or(r1, And(LShift(r1, 17), And(Not(pos),mask2)));
      r3 = Or(r3, And(LShift(r2, 17), And(Not(pos),mask2)));
      r4 = Or(r4, And(LShift(r3, 17), And(Not(pos),mask2)));
      r5 = Or(r5, And(LShift(r4, 17), And(Not(pos),mask2)));
      r6 = Or(r6, And(LShift(r5, 17), And(Not(pos),mask2)));
      r7 = Or(r7, And(LShift(r6, 17), And(Not(pos),mask2)));
      r8 = Or(r8, And(LShift(r7, 17), And(Not(pos),mask2)));
      r9 = Or(r9, And(LShift(r8, 17), And(Not(pos),mask2)));
      r10 = Or(r10, And(LShift(r9, 17), And(Not(pos),mask2)));
      r11 = Or(r11, And(LShift(r10, 17), And(Not(pos),mask2)));
      r12 = Or(r12, And(LShift(r11, 17), And(Not(pos),mask2)));
      r13 = Or(r13, And(LShift(r12, 17), And(Not(pos),mask2)));
      r14 = Or(r14, And(LShift(r13, 17), And(Not(pos),mask2)));
      r15 = Or(r15, And(LShift(r14, 17), And(Not(pos),mask2)));
      r16 = Or(r16, And(LShift(r15, 17), And(Not(pos),mask2)));
  

      l1 = Or(l1, And(RShift(queens,17), And(Not(pos),mask2)));
      l2 = Or(l1, And(RShift(l1,17), And(Not(pos),mask2)));
      l3 = Or(l3, And(RShift(l2,17), And(Not(pos),mask2)));
      l4 = Or(l4, And(RShift(l3,17), And(Not(pos),mask2)));
      l5 = Or(l5, And(RShift(l4,17), And(Not(pos),mask2)));
      l6 = Or(l6, And(RShift(l5,17), And(Not(pos),mask2)));
      l7 = Or(l7, And(RShift(l6,17), And(Not(pos),mask2)));
      l8 = Or(l8, And(RShift(l7,17), And(Not(pos),mask2)));
      l9 = Or(l9, And(RShift(l8,17), And(Not(pos),mask2)));
      l10 = Or(l10, And(RShift(l9,17), And(Not(pos),mask2)));
      l11 = Or(l11, And(RShift(l10,17), And(Not(pos),mask2)));
      l12 = Or(l12, And(RShift(l11,17), And(Not(pos),mask2)));
      l13 = Or(l13, And(RShift(l12,17), And(Not(pos),mask2)));
      l14 = Or(l14, And(RShift(l13,17), And(Not(pos),mask2)));
      l15 = Or(l15, And(RShift(l14,17), And(Not(pos),mask2)));
      l16 = Or(l16, And(RShift(l15,17), And(Not(pos),mask2)));

    }
    
    if(Positive(And(queens, mask1))) {

      r1 = Or(r1, And(LShift(queens, 17), And(Not(pos),mask1)));
      r2 = Or(r2, And(LShift(r1, 17), And(Not(pos),mask1)));
      r3 = Or(r3, And(LShift(r2, 17), And(Not(pos),mask1)));
      r4 = Or(r4, And(LShift(r3, 17), And(Not(pos),mask1)));
      r5 = Or(r5, And(LShift(r4, 17), And(Not(pos),mask1)));
      r6 = Or(r6, And(LShift(r5, 17), And(Not(pos),mask1)));
      r7 = Or(r7, And(LShift(r6, 17), And(Not(pos),mask1)));
      r8 = Or(r8, And(LShift(r7, 17), And(Not(pos),mask1)));
      r9 = Or(r9, And(LShift(r8, 17), And(Not(pos),mask1)));
      r10 = Or(r10, And(LShift(r9, 17), And(Not(pos),mask1)));
      r11 = Or(r11, And(LShift(r10, 17), And(Not(pos),mask1)));
      r12 = Or(r12, And(LShift(r11, 17), And(Not(pos),mask1)));
      r13 = Or(r13, And(LShift(r12, 17), And(Not(pos),mask1)));
      r14 = Or(r14, And(LShift(r13, 17), And(Not(pos),mask1)));
      r15 = Or(r15, And(LShift(r14, 17), And(Not(pos),mask1)));
      r16 = Or(r16, And(LShift(r15, 17), And(Not(pos),mask1)));
  

      l1 = Or(l1, And(RShift(queens,17), And(Not(pos),mask1)));
      l2 = Or(l2, And(RShift(l1,17), And(Not(pos),mask1)));
      l3 = Or(l3, And(RShift(l2,17), And(Not(pos),mask1)));
      l4 = Or(l4, And(RShift(l3,17), And(Not(pos),mask1)));
      l5 = Or(l5, And(RShift(l4,17), And(Not(pos),mask1)));
      l6 = Or(l6, And(RShift(l5,17), And(Not(pos),mask1)));
      l7 = Or(l7, And(RShift(l6,17), And(Not(pos),mask1)));
      l8 = Or(l8, And(RShift(l7,17), And(Not(pos),mask1)));
      l9 = Or(l9, And(RShift(l8,17), And(Not(pos),mask1)));
      l10 = Or(l10, And(RShift(l9,17), And(Not(pos),mask1)));
      l11 = Or(l11, And(RShift(l10,17), And(Not(pos),mask1)));
      l12 = Or(l12, And(RShift(l11,17), And(Not(pos),mask1)));
      l13 = Or(l13, And(RShift(l12,17), And(Not(pos),mask1)));
      l14 = Or(l14, And(RShift(l13,17), And(Not(pos),mask1)));
      l15 = Or(l15, And(RShift(l14,17), And(Not(pos),mask1)));
      l16 = Or(l16, And(RShift(l15,17), And(Not(pos),mask1)));

     
    }
      
  } 
  
  for(MBOARD mask1 = BISHOP2, mask2 = BISHOP2; Positive(mask2); mask1 = RShiftBishop1(mask1,1), mask2 = LShiftBishop1(mask2,1) ) {
     if(Positive(And(queens, mask1))) {
      u1 = Or(u1, And(LShift(queens,15), And(Not(pos),mask1)));
      u2 = Or(u2, And(LShift(u1,15), And(Not(pos),mask1)));
      u3 = Or(u3, And(LShift(u2,15), And(Not(pos),mask1)));
      u4 = Or(u4, And(LShift(u3,15), And(Not(pos),mask1)));
      u5 = Or(u5, And(LShift(u4,15), And(Not(pos),mask1)));
      u6 = Or(u6, And(LShift(u5,15), And(Not(pos),mask1)));
      u7 = Or(u7, And(LShift(u6,15), And(Not(pos),mask1)));
      u8 = Or(u8, And(LShift(u7,15), And(Not(pos),mask1)));
      u9 = Or(u9, And(LShift(u8,15), And(Not(pos),mask1)));
      u10 = Or(u10, And(LShift(u9,15), And(Not(pos),mask1)));
      u11 = Or(u11, And(LShift(u10,15), And(Not(pos),mask1)));
      u12 = Or(u12, And(LShift(u11,15), And(Not(pos),mask1)));
      u13 = Or(u13, And(LShift(u12,15), And(Not(pos),mask1)));
      u14 = Or(u14, And(LShift(u13,15), And(Not(pos),mask1)));
      u15 = Or(u15, And(LShift(u14,15), And(Not(pos),mask1)));
      u16 = Or(u16, And(LShift(u15,15), And(Not(pos),mask1)));

      d1 = Or(d1, And(RShift(queens,15), And(Not(pos),mask1)));
      d2 = Or(d2, And(RShift(d1,15), And(Not(pos),mask1)));
      d3 = Or(d3, And(RShift(d2,15), And(Not(pos),mask1)));
      d4 = Or(d4, And(RShift(d3,15), And(Not(pos),mask1)));
      d5 = Or(d5, And(RShift(d4,15), And(Not(pos),mask1)));
      d6 = Or(d6, And(RShift(d5,15), And(Not(pos),mask1)));
      d7 = Or(d7, And(RShift(d6,15), And(Not(pos),mask1)));
      d8 = Or(d8, And(LShift(d7,15), And(Not(pos),mask1)));
      d9 = Or(d9, And(RShift(d8,15), And(Not(pos),mask1)));
      d10 = Or(d10, And(RShift(d9,15), And(Not(pos),mask1)));
      d11 = Or(d11, And(RShift(d10,15), And(Not(pos),mask1)));
      d12 = Or(d12, And(RShift(d11,15), And(Not(pos),mask1)));
      d13 = Or(d13, And(RShift(d12,15), And(Not(pos),mask1)));
      d14 = Or(d14, And(RShift(d13,15), And(Not(pos),mask1)));
      d15 = Or(d15, And(RShift(d14,15), And(Not(pos),mask1)));
      d16 = Or(d16, And(RShift(d15,15), And(Not(pos),mask1)));
    }
    if(Positive(And(queens,mask2))) {
      u1 = Or(u1, And(LShift(queens,15), And(Not(pos),mask2)));
      u2 = Or(u2, And(LShift(u1,15), And(Not(pos),mask2)));
      u3 = Or(u3, And(LShift(u2,15), And(Not(pos),mask2)));
      u4 = Or(u4, And(LShift(u3,15), And(Not(pos),mask2)));
      u5 = Or(u5, And(LShift(u4,15), And(Not(pos),mask2)));
      u6 = Or(u6, And(LShift(u5,15), And(Not(pos),mask2)));
      u7 = Or(u7, And(LShift(u6,15), And(Not(pos),mask2)));
      u8 = Or(u8, And(LShift(u7,15), And(Not(pos),mask2)));
      u9 = Or(u9, And(LShift(u8,15), And(Not(pos),mask2)));
      u10 = Or(u10, And(LShift(u9,15), And(Not(pos),mask2)));
      u11 = Or(u11, And(LShift(u10,15), And(Not(pos),mask2)));
      u12 = Or(u12, And(LShift(u11,15), And(Not(pos),mask2)));
      u13 = Or(u13, And(LShift(u12,15), And(Not(pos),mask2)));
      u14 = Or(u14, And(LShift(u13,15), And(Not(pos),mask2)));
      u15 = Or(u15, And(LShift(u14,15), And(Not(pos),mask2)));
      u16 = Or(u16, And(LShift(u15,15), And(Not(pos),mask2)));

      d1 = Or(d1, And(RShift(queens,15), And(Not(pos),mask2)));
      d2 = Or(d1, And(RShift(d1,15), And(Not(pos),mask2)));
      d3 = Or(d2, And(RShift(d2,15), And(Not(pos),mask2)));
      d4 = Or(d3, And(RShift(d3,15), And(Not(pos),mask2)));
      d5 = Or(d4, And(RShift(d4,15), And(Not(pos),mask2)));
      d6 = Or(d5, And(RShift(d5,15), And(Not(pos),mask2)));
      d7 = Or(d7, And(RShift(d6,15), And(Not(pos),mask2)));
      d8 = Or(d8, And(LShift(d7,15), And(Not(pos),mask2)));
      d9 = Or(d9, And(RShift(d8,15), And(Not(pos),mask2)));
      d10 = Or(d10, And(RShift(d9,15), And(Not(pos),mask2)));
      d11 = Or(d11, And(RShift(d10,15), And(Not(pos),mask2)));
      d12 = Or(d12, And(RShift(d11,15), And(Not(pos),mask2)));
      d13 = Or(d13, And(RShift(d12,15), And(Not(pos),mask2)));
      d14 = Or(d14, And(RShift(d13,15), And(Not(pos),mask2)));
      d15 = Or(d15, And(RShift(d14,15), And(Not(pos),mask2)));
      d16 = Or(d16, And(RShift(d15,15), And(Not(pos),mask2)));
      }   
  }
  
  MBOARD bishopMask1 =  Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(r1, r2),r3),r4),r5),r6),r7),r8),l1),
											     l2),l3),l4),l5), l6), l7), l8),
									u1), u2), u3), u4), u5), u6), u7), u8),d1), d2), d3), d4), d5), d6), d7), d8);

  MBOARD bishopMask2 =  Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(Or(r9, r10),r11),r12),r13),r14),r15),r16),l9),
											     l10),l11),l12),l13), l14), l15), l16),
									u9), u10), u11), u12), u13), u14), u15), u16),d9), d10), d11), d12), d13), d14), d15), d16);
  
  return Or(bishopMask1, bishopMask2);
  } 

int main() {
  MBOARD B;
  B.board[0] = 0xFFFFULL;
  B.board[1] = 0xFFFFULL;
  B.board[2] = 0xFFFFULL;
  B.board[3] = 0xFFFFULL;
  drawBoard(B,compliment(B));
  MBOARD C = {.board = {1,1,1,1}};
  drawBoard(C,compliment(C));
  getBit(compliment(B),34);
  printf("get bit %llu \n", (1ULL << 33) & compliment(B).board[0]);
  MBOARD MM = {.board = {1ULL << 60,1,1,1}};
  drawBoard(MM,MM);
  drawBoard(RShift(MM,8),MM);
  drawBoard(bishopDiagonal1(),RShiftBishop1(bishopDiagonal1(),3));
  MBOARD rook = {.board = {0xFFFF,0,0,0}};
  drawBoard(LShiftRook(rook,5),{0});
  drawBoard(getBishopMask(UShiftRook(C,5)),UShiftRook(C,5));
  return 0;
}
